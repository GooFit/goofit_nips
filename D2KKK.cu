#include "hip/hip_runtime.h"
// ROOT stuff
#include <TApplication.h>
#include <TCanvas.h>
#include <TFile.h>
#include <TH1F.h>
#include <TH2F.h>
#include <TGraph.h>
#include <TLegend.h>
#include <TMath.h>
#include <TRandom.h>
#include <TRandom3.h>
#include <TTree.h>
#include <TROOT.h>
#include <TMinuit.h>


// System stuff
#include <CLI/Timer.hpp>
#include <fstream>


// GooFit stuff
#include <goofit/Application.h>
#include <goofit/BinnedDataSet.h>
#include <goofit/FitManager.h>
#include <goofit/fitting/FitManagerMinuit2.h>
#include <goofit/PDFs/GooPdf.h>
#include <goofit/PDFs/basic/PolynomialPdf.h>
#include <goofit/PDFs/basic/SmoothHistogramPdf.h>
#include <goofit/PDFs/combine/AddPdf.h>
#include <goofit/PDFs/combine/ProdPdf.h>
#include <goofit/PDFs/physics/DalitzPlotPdf.h>
#include <goofit/PDFs/physics/DalitzVetoPdf.h>
#include <goofit/PDFs/physics/ResonancePdf.h>
#include <goofit/UnbinnedDataSet.h>
#include <goofit/Variable.h>

#include <goofit/utilities/Style.h>
#include <goofit/PDFs/physics/DalitzPlotter.h>

#include <Eigen/Dense>

#include <Minuit2/MnScan.h>
#include <TNtuple.h>

#include <thrust/transform_reduce.h>

using namespace std;
using namespace GooFit;
using namespace ROOT;

UnbinnedDataSet *data    = nullptr;

TH2F *weightHistogram    = nullptr;
TH2F *bkgHistogram       = nullptr;
TH2F *underlyingBins     = nullptr;

// How many events will be generated for Eff Bkg?
const double NevG = 1e7;

// PWA INPUT FILE NAME
const string pwa_file = "files/PWA_COEFFS_50.txt";

// FIT OR JUST PLOT?
bool fit = true;

const fptype _mDp      = 1.86962; //D mass
//const fptype _mDp      = 5.27932;
const fptype KPlusMass = 0.493677; //K^{+/-} mass

const fptype D1Mass  = KPlusMass; //Daughter 1 Mass K^{-}
const fptype D2Mass  = KPlusMass; //Daughter 2 Mass K^{+}
const fptype D3Mass  = KPlusMass; //Daughter 3 Mass K^{+}make

const fptype D1Mass2 = D1Mass * D1Mass;
const fptype D2Mass2 = D2Mass * D2Mass;
const fptype D3Mass2 = D3Mass * D3Mass;
const fptype MMass   = _mDp; //Mother Mass
const fptype MMass2  = MMass * MMass;

fptype m12_min = pow(D1Mass  + D2Mass,2);
fptype m12_max = pow(_mDp - D2Mass,2);
fptype m13_min = pow(D1Mass  + D3Mass,2);
fptype m13_max = pow(_mDp - D3Mass,2);


Observable m12("m12", m12_min, m12_max);
Observable m13("m13", m13_min, m13_max);

EventNumber eventNumber("eventNumber");
bool fitMasses = false;


// Constants used in more than one PDF component
Variable motherM("motherM", MMass);
Variable massSum("massSum", MMass2 + D1Mass2 + D2Mass2 + D3Mass2); // = 3.53481
Variable constantOne("constantOne", 1);
Variable constantZero("constantZero", 0);

//global vector of pdf components
std::vector<PdfBase *> comps;

double V = (m12.getUpperLimit() - m12.getLowerLimit()) * (m13.getUpperLimit() - m13.getLowerLimit());

int verbosity = 3;

GooPdf *kzero_veto = nullptr;
double mesonRad = 1.5;

DalitzPlotPdf *signalDalitz = nullptr;

bool doEffSwap   = true;
bool saveEffPlot = true;
bool saveBkgPlot = true;

vector<fptype> HH_bin_limits;
vector<Variable> pwa_coefs_amp;
vector<Variable> pwa_coefs_phs;

DalitzPlotPdf *makeSignalPdf(GooPdf *eff = 0, bool fixAmps = false);

fptype cpuGetM23(fptype massPZ, fptype massPM) { return (massSum.getValue() - massPZ - massPM); }

void makeToyDalitzData(GooPdf *overallSignal, std::string name, size_t nTotal) {

    DalitzPlotter dp(overallSignal, signalDalitz);


    // Generate data
    data = new UnbinnedDataSet({m12, m13, eventNumber});

    { // Plotting block
        TCanvas foo;
        auto th1 = dp.make2D();
        th1->Rebin2D(5,5);
        th1->Draw("COLZ");
        foo.SaveAs("plots/plot1.png");
    }

    dp.fillDataSetMC(*data, nTotal);

    TH2F dalitzpp0_dat_hist("dalitzpp0_dat_hist",
                            "",
                            200,
                            m12.getLowerLimit(),
                            m12.getUpperLimit(),
                            200,
                            m13.getLowerLimit(),
                            m13.getUpperLimit());
    dalitzpp0_dat_hist.GetXaxis()->SetTitle("m^{2}(K^{-}K^{+}) [GeV^{2}]");
    dalitzpp0_dat_hist.GetYaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV^{2}]");

    // Make a writer
    {
        ofstream writer(name);

        // Fill histogram with generated data
        for(size_t i=0; i<data->getNumEvents(); i++) {
            data->loadEvent(i);
            dalitzpp0_dat_hist.Fill(m12, m13);

            writer << i << '\t' << m12.getValue() << '\t' << m13.getValue() << '\n';
        }
    }



    std::cout << "Entries generated: " << data->getNumEvents() << std::endl;
    TCanvas foo;
    foo.SetLogz(false);
    dalitzpp0_dat_hist.Draw("colz");
    dalitzpp0_dat_hist.SetStats(0);
    foo.SaveAs("plots/Dalitz_D2KKK_temp.png");
}

void runToyGeneration(std::string name, size_t events) {
    m12.setNumBins(1500);
    m13.setNumBins(1500);

    signalDalitz = makeSignalPdf(0, false);

    std::cout << "Creating overall PDF\n";
    ProdPdf *overallSignal = new ProdPdf("overallSignal", {signalDalitz});

    {
        makeToyDalitzData(overallSignal, name, events);
    }

}

void getToyData(std::string toyFileName, bool info) {
    TH2F dalitzplot("dalitzplot",
                    "",
                    m12.getNumBins(),
                    m12.getLowerLimit(),
                    m12.getUpperLimit(),
                    m13.getNumBins(),
                    m13.getLowerLimit(),
                    m13.getUpperLimit());

    data = new UnbinnedDataSet({m12, m13, eventNumber});

    const string suffix = ".root";

    if(toyFileName.rfind(suffix) + suffix.length() == toyFileName.length()) {
        GOOFIT_INFO("Reading ROOT file: {}", toyFileName);

        TFile *f = TFile::Open(toyFileName.c_str());
        TTree *t = (TTree *)f->Get("DecayTree");

        std::cout << "Entries: " << t->GetEntries() << std::endl;
        assert(t);
        double m2_12, m2_13;

        t->SetBranchAddress("s12", &m2_12);
        t->SetBranchAddress("s13", &m2_13);
        for(int i = 0; i < t->GetEntries(); i++) {

            t->GetEntry(i);
            m12.setValue(m2_12);
            m13.setValue(m2_13);
            eventNumber.setValue(data->getNumEvents());
            data->addEvent();
            dalitzplot.Fill(m12.getValue(), m13.getValue());
        }
        f->Close();
    } else {
        if(info == true) {
            GOOFIT_INFO("Reading 3 column TEXT file: {}", toyFileName);
        }
        std::ifstream reader(toyFileName.c_str());

        while(reader >> eventNumber >> m12 >> m13) {
            data->addEvent();
            dalitzplot.Fill(m12.getValue(), m13.getValue());
        }
    }

    if(info == true){
    TCanvas foo;
    dalitzplot.SetStats(0);
    dalitzplot.Draw("colz");
    foo.SaveAs("plots/dalitzplot_D2KKK_gen.png");}
}

void createWeightHistogram() {
    TFile *f        = TFile::Open("files/effspline300.root");
    weightHistogram = (TH2F *)f->Get("eff_spline");
    weightHistogram->SetStats(false);
}

void createBackgroundHistogram() {
    TFile *f     = TFile::Open("files/bkg_histo_300bins.root");
    bkgHistogram = (TH2F *)f->Get("bkgHist_acc");
    bkgHistogram->SetStats(false);
}

GooPdf *makeEfficiencyPdf() {

    vector<Observable> lvars;
    lvars.push_back(m12);
    lvars.push_back(m13);
    BinnedDataSet *binEffData = new BinnedDataSet(lvars);
    createWeightHistogram();

    TRandom3 donram(0);
    for(int i = 0; i < NevG; i++) {
        do {
            m12.setValue(donram.Uniform(m12.getLowerLimit(), m12.getUpperLimit()));
            m13.setValue(donram.Uniform(m13.getLowerLimit(), m13.getUpperLimit()));
        } while(!inDalitz(m12.getValue(), m13.getValue(), MMass, D1Mass, D2Mass, D3Mass));

        double weight = weightHistogram->GetBinContent(weightHistogram->FindBin(m12.getValue(), m13.getValue()));
        binEffData->addWeightedEvent(weight);

        if(doEffSwap) {
            double swapmass = m12.getValue();
            m12.setValue(m13.getValue());
            m13.setValue(swapmass);
            weight = weightHistogram->GetBinContent(weightHistogram->FindBin(m12.getValue(), m13.getValue()));
            binEffData->addWeightedEvent(weight);
        }
    }
    if(saveEffPlot) {
        TCanvas foo;
        foo.cd();
        weightHistogram->Draw("colz");
        foo.SaveAs("plots/efficiency_bins.png");
        foo.SetLogz(true);
        foo.SaveAs("plots/efficiency_bins_log.png");
    }
    // Smooth
    Variable effSmoothing("effSmoothing", 0);
    SmoothHistogramPdf *ret = new SmoothHistogramPdf("efficiency", binEffData, effSmoothing);
    return ret;
}

GooPdf *makeBackgroundPdf() {

    BinnedDataSet *binBkgData = new BinnedDataSet({m12, m13});
    createBackgroundHistogram();

    TRandom3 donram(0);
    for(int i = 0; i < NevG; i++) {
        do {
            m12.setValue(donram.Uniform(m12.getLowerLimit(), m12.getUpperLimit()));
            m13.setValue(donram.Uniform(m13.getLowerLimit(), m13.getUpperLimit()));
        } while(!inDalitz(m12.getValue(), m13.getValue(), MMass, D1Mass, D2Mass, D3Mass));

        double weight = bkgHistogram->GetBinContent(bkgHistogram->FindBin(m12.getValue(), m13.getValue()));
        binBkgData->addWeightedEvent(weight);

        if(doEffSwap) {
            double swapmass = m12.getValue();
            m12.setValue(m13.getValue());
            m13.setValue(swapmass);
            weight = bkgHistogram->GetBinContent(bkgHistogram->FindBin(m12.getValue(), m13.getValue()));
            binBkgData->addWeightedEvent(weight);
        }
    }
    if(saveBkgPlot) {
        TCanvas foo;
        bkgHistogram->Draw("colz");
        foo.SetLogz(false);
        foo.SaveAs("plots/background_bins.png");
        foo.SetLogz(true);
        foo.SaveAs("plots/background_bins_log.png");
    }
    Variable *effSmoothing  = new Variable("effSmoothing", 0);
    SmoothHistogramPdf *ret = new SmoothHistogramPdf("efficiency", binBkgData, *effSmoothing);
    return ret;
}

ResonancePdf *loadPWAResonance(const string fname = pwa_file, bool fixAmp = false) {

    std::ifstream reader;
	//GOOFIT_INFO("LOADING FILE {}",fname);
    reader.open(fname.c_str());
    assert(reader.good());
    HH_bin_limits.clear();
    pwa_coefs_amp.clear();
    pwa_coefs_phs.clear();

    double e1, e2, e3, e4;
    double emag, ephs;
    int i = 0;
    while(reader >> e1 >> e2 >> e3 >> e4) {

        HH_bin_limits.push_back(e1 * e1);

        emag = sqrt(e2 * e2 + e3 * e3);
        ephs = TMath::ATan2(e3, e2);

        Variable va(fmt::format("pwa_coef_{}_mag", i), emag, .000001, 0, 10000);
        Variable vp(fmt::format("pwa_coef_{}_phase", i), ephs, .000001, -360, 360);

        pwa_coefs_amp.push_back(va);
        pwa_coefs_phs.push_back(vp);
        i++;

    }

    Variable swave_amp_real("swave_amp_real", 3.0, 0.001, 0, 0);
    Variable swave_amp_imag("swave_amp_imag", 0.0, 0.001, 0, 0);
    swave_amp_real.setFixed(true);
    swave_amp_imag.setFixed(true);

    if(fixAmp) {
        swave_amp_real.setValue(1.);
        swave_amp_imag.setValue(0.);
        swave_amp_real.setFixed(true);
        swave_amp_imag.setFixed(true);
    }
    cout << "Numbers loaded: " << HH_bin_limits.size() << " / " << i << endl;

    ResonancePdf *swave_12 = new Resonances::Spline(
        "swave_12", swave_amp_real, swave_amp_imag, HH_bin_limits, pwa_coefs_amp, pwa_coefs_phs, PAIR_12, true);

    return swave_12;
}

DalitzPlotPdf *makeSignalPdf(GooPdf *eff, bool fixAmps) {

    DecayInfo3 dtop0pp;
    dtop0pp.motherMass   = MMass;
    dtop0pp.daug1Mass    = D1Mass;
    dtop0pp.daug2Mass    = D2Mass;
    dtop0pp.daug3Mass    = D3Mass;
    dtop0pp.meson_radius = 1.5;

    // phi

	Variable fixedPhiMass("phi_mass", 1.019461, 0.01, 0.7, 1.8);
	Variable fixedPhiWidth("phi_width", 0.004266, 0.001, 1e-5, 1e-1);
    Variable phi_amp_real("phi_amp_real", 1);
    Variable phi_amp_imag("phi_amp_imag", 0);
    fixedPhiMass.setFixed(true);
    fixedPhiWidth.setFixed(true);

    ResonancePdf *phi
        = new Resonances::RBW("phi", phi_amp_real, phi_amp_imag, fixedPhiMass, fixedPhiWidth, 1, PAIR_12, true); //(true) Required to be symmetric

    // f0(980)

	Variable f0_amp_real("f0_amp_real", 12.341 * cos(-62.852 * (M_PI / 180)), 0.0001, -100, 100);
	Variable f0_amp_imag("f0_amp_imag", 12.341 * sin(-62.852 * (M_PI / 180)), 0.0001, -100, 100);
    Variable f0Mass("f0Mass", 0.965);
    Variable f0g1("f0g1", 0.165);
    Variable rg1og2("rg1og2", 4.21*0.165);

    ResonancePdf *f0
            = new Resonances::FLATTE("f0", f0_amp_real, f0_amp_imag, f0Mass, f0g1, rg1og2, PAIR_12, true); //(true) Required to be symmetric

    // f0(X) Mass Fitted From Data

    Variable f0X_amp_real("f0X_amp_real", 11.918 * cos(20.248 * (M_PI / 180)), 0.0001, -100, 100);
    Variable f0X_amp_imag("f0X_amp_imag", 11.918 * sin(20.248 * (M_PI / 180)), 0.0001, -100, 100);
    Variable f0XMass("f0XMass", 1.41478);
    Variable f0XWidth("f0XWidth", 0.309491);

    ResonancePdf *f0X = new Resonances::RBW("f0X", f0X_amp_real, f0X_amp_imag, f0XMass, f0XWidth, (unsigned int)0, PAIR_12, true); //(true) Required to be symmetric

    // NR
    Variable nonr_amp_real("nonr_amp_real", 1.0, 0.001, -100, +100);
    Variable nonr_amp_imag("nonr_amp_imag", 0.0, 0.001, -100, +100);
    ResonancePdf *nonr = new Resonances::NonRes("nonr", nonr_amp_real, nonr_amp_imag);

    //PWA
    ResonancePdf *swave_12 = loadPWAResonance(pwa_file, fixAmps);

    //Adding resonances

    dtop0pp.resonances.push_back(phi);
    dtop0pp.resonances.push_back(f0X);
    dtop0pp.resonances.push_back(f0);
    //dtop0pp.resonances.push_back(nonr);


    //PWA analysis
    //dtop0pp.resonances.push_back(swave_12);


    if(!eff) {
        // By default create a constant efficiency.
        vector<Variable> offsets;
        vector<Observable> observables;
        vector<Variable> coefficients;

        observables.push_back(m12);
        observables.push_back(m13);
        offsets.push_back(constantZero);
        offsets.push_back(constantZero);
        coefficients.push_back(constantOne);
        eff = new PolynomialPdf("constantEff", observables, coefficients, offsets, 0); //No efficiency
    }
    comps.clear();

    return new DalitzPlotPdf("signalPDF", m12, m13, eventNumber, dtop0pp, eff);
}

std::tuple<double, double> DalitzNorm(GooPdf *overallSignal, int N) { //Brute Force integration, expected integral value = 1.0
    random_device rd;
    mt19937 mt(rd());
    uniform_real_distribution<double> xyvalues(m12.getLowerLimit(), m12.getUpperLimit());

    std::vector<Observable> vars;
    vars.push_back(m12);
    vars.push_back(m13);
    vars.push_back(eventNumber);

    std::vector<fptype> rpdfValuesvec;

    UnbinnedDataSet data(vars);
    eventNumber = 0;

    for(int i = 0; i < N; i++) {
        m12 = xyvalues(mt);
        m13 = xyvalues(mt);

        if(inDalitz(m12.getValue(), m13.getValue(), MMass, D1Mass, D2Mass, D3Mass) == 1) {
            eventNumber.setValue(eventNumber.getValue() + 1);
            data.addEvent();
        }
    }

    overallSignal->setData(&data);
    signalDalitz->setDataSize(data.getNumEvents());

    std::vector<std::vector<double>> pdfValues = overallSignal->getCompProbsAtDataPoints();

    double buffer = 0;

    for(int k = 0; k < pdfValues[0].size(); k++) {
        buffer += pdfValues[0][k];
    }

    double mean = buffer / N; //sum
    double diff = 0;

    for(int l = 0; l < pdfValues[0].size(); l++) {
        diff += (pdfValues[0][l] - mean) * (pdfValues[0][l] - mean);
    }

    double variance_f = diff / (N - 1);
    double variance   = V * V * variance_f / N;
    double sigma      = sqrt(variance);
    double integral   = V * mean;

    return std::make_tuple(integral, sigma);
}

void runIntegration(int N ,int Nint) {

    signalDalitz = makeSignalPdf(0, false);
    std::vector<PdfBase *> comps;
    comps.clear();
    comps.push_back(signalDalitz);
    ProdPdf *overallSignal = new ProdPdf("overallSignal", comps);

    double arr[Nint];
    double arr_error[Nint];
    std::fill(arr, arr + Nint, 0);
    std::fill(arr_error, arr_error + Nint, 0);

    TH1D integral_hist("integral", "integral", 30, arr[0] * (0.95), arr[Nint-1] * (1.05));

    for(int i = 0; i < Nint; i++) {
        auto integral2 = DalitzNorm(overallSignal, N);
        arr[i]         = std::get<0>(integral2);
        arr_error[i]   = std::get<1>(integral2);
    }

    std::sort(arr, arr + Nint);

    for(int l = 0; l < Nint; l++) {
        integral_hist.Fill(arr[l]);
    }

    integral_hist.GetXaxis()->SetTitle("Integral");
    integral_hist.GetYaxis()->SetTitle("Frequency");

    TCanvas integral_Canvas("integral", "integral", 800, 800);
    integral_hist.Draw("E");
    integral_Canvas.SaveAs("plots/D2KKK_Plots_Integral.png");

    std::cout << '\n';
    std::cout << "<E>_{N_Integrations=100}: " << integral_hist.GetMean() << '\t'
              << "stdError: " << integral_hist.GetMeanError() << "\t\t"
              << "stdDev: " << integral_hist.GetStdDev() << "\n\n";

    double integral, sigma2;
    std::tie(integral, sigma2) = DalitzNorm(overallSignal, N);

    std::cout << "<E>_{N_Integrations=1}: " << integral << '\t' << "<delta_E>: " << sigma2 << "\n\n";
    std::cout << "|stdDev - <delta_E>|= " << abs(integral_hist.GetStdDev() - sigma2) << "\n\n";

}

void drawFitPlotsWithPulls(TH1 *hd, TH1 *ht, string plotdir) {
    const char *hname = hd->GetName();
    char obsname[10];
    for(int i = 0;; i++) {
        if(hname[i] == '_')
            obsname[i] = '\0';
        else
            obsname[i] = hname[i];
        if(obsname[i] == '\0')
            break;
    }
    ht->Scale(hd->Integral() / ht->Integral()*5);
	ht->SetLineColor(kRed);
    ht->SetLineWidth(3);
    ht->SetMarkerStyle(0);

	hd->SetMarkerColor(kBlack);
	hd->Rebin(5);


    TCanvas foo;

	hd->Draw("E");
    ht->Draw("HIST C same");


    foo.SaveAs(TString::Format("plots/%s_fit.png",obsname));


}


void makeToyDalitzPdfPlots(GooPdf *overallSignal, string plotdir = "plots") {
    TH1F m12_dat_hist("m12_dat_hist", "", m12.getNumBins(), m12.getLowerLimit(), m12.getUpperLimit());
    m12_dat_hist.GetXaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV]");
    m12_dat_hist.GetYaxis()->SetTitle(TString::Format("Events / %.1f MeV", 1e3 * m12_dat_hist.GetBinWidth(1)));

    TH1F m12_pdf_hist("m12_pdf_hist", "", m12.getNumBins(), m12.getLowerLimit(), m12.getUpperLimit());

    TH1F m13_dat_hist("m13_dat_hist", "", m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());
    m13_dat_hist.GetXaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV]");
    m13_dat_hist.GetYaxis()->SetTitle(TString::Format("Events / %.1f MeV", 1e3 * m13_dat_hist.GetBinWidth(1)));

    TH1F m13_pdf_hist("m13_pdf_hist", "", m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());

    TH1F m23_dat_hist("m23_dat_hist", "", m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());
    m23_dat_hist.GetXaxis()->SetTitle("m^{2}(#pi^{+} #pi^{-}) [GeV]");
    m23_dat_hist.GetYaxis()->SetTitle(TString::Format("Events / %.1f MeV", 1e3 * m13_dat_hist.GetBinWidth(1)));

    TH1F m23_pdf_hist("m23_pdf_hist", "", m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());

    double totalPdf = 0;
    double totalDat = 0;
    TH2F dalitzpp0_dat_hist("dalitzpp0_dat_hist",
                            "",
                            m12.getNumBins(),
                            m12.getLowerLimit(),
                            m12.getUpperLimit(),
                            m13.getNumBins(),
                            m13.getLowerLimit(),
                            m13.getUpperLimit());
    dalitzpp0_dat_hist.SetStats(false);
    dalitzpp0_dat_hist.GetXaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV]");
    dalitzpp0_dat_hist.GetYaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV^{2}]");
    TH2F dalitzpp0_pdf_hist("dalitzpp0_pdf_hist",
                            "",
                            m12.getNumBins(),
                            m12.getLowerLimit(),
                            m12.getUpperLimit(),
                            m13.getNumBins(),
                            m13.getLowerLimit(),
                            m13.getUpperLimit());

    dalitzpp0_pdf_hist.GetXaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV^{2}]");
    dalitzpp0_pdf_hist.GetYaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV^{2}]");
    dalitzpp0_pdf_hist.SetStats(false);
    std::vector<Observable> vars;
    vars.push_back(m12);
    vars.push_back(m13);
    vars.push_back(eventNumber);
    UnbinnedDataSet currData(vars);
    int evtCounter = 0;

    for(int i = 0; i < m12.getNumBins(); ++i) {
        m12.setValue(m12.getLowerLimit() + (m12.getUpperLimit() - m12.getLowerLimit()) * (i + 0.5) / m12.getNumBins());
        for(int j = 0; j < m13.getNumBins(); ++j) {
            m13.setValue(m13.getLowerLimit()
                         + (m13.getUpperLimit() - m13.getLowerLimit()) * (j + 0.5) / m13.getNumBins());
            if(!inDalitz(m12.getValue(), m13.getValue(), MMass, D1Mass, D2Mass, D3Mass))
                continue;
            eventNumber.setValue(evtCounter);
            evtCounter++;
            currData.addEvent();
        }
    }
    overallSignal->setData(&currData);
    signalDalitz->setDataSize(currData.getNumEvents());
    std::vector<std::vector<double>> pdfValues = overallSignal->getCompProbsAtDataPoints();
    for(unsigned int j = 0; j < pdfValues[0].size(); ++j) {
        double currm12 = currData.getValue(m12, j);
        double currm13 = currData.getValue(m13, j);

        dalitzpp0_pdf_hist.Fill(currm12, currm13, pdfValues[0][j]);
        m12_pdf_hist.Fill(currm12, pdfValues[0][j]);
        m13_pdf_hist.Fill(currm13, pdfValues[0][j]);
        m23_pdf_hist.Fill(cpuGetM23(currm12, currm13), pdfValues[0][j]);
        totalPdf += pdfValues[0][j];
    }

    TCanvas foo;
    foo.SetLogz(false);
    dalitzpp0_pdf_hist.Draw("colz");

    foo.SaveAs("plots/dalitzpp0_pdf.png");

    for(unsigned int evt = 0; evt < data->getNumEvents(); ++evt) {
        double data_m12 = data->getValue(m12, evt);
        m12_dat_hist.Fill(data_m12);
        double data_m13 = data->getValue(m13, evt);
        m13_dat_hist.Fill(data_m13);
        dalitzpp0_dat_hist.Fill(data_m12, data_m13);
        m23_dat_hist.Fill(cpuGetM23(data_m12, data_m13));
        totalDat++;
    }
    dalitzpp0_dat_hist.Draw("colz");
    foo.SaveAs("plots/dalitzpp0_dat.png");

    drawFitPlotsWithPulls(&m12_dat_hist, &m12_pdf_hist, plotdir);
    drawFitPlotsWithPulls(&m13_dat_hist, &m13_pdf_hist, plotdir);
    drawFitPlotsWithPulls(&m23_dat_hist, &m23_pdf_hist, plotdir);
}



void PrintFF(std::vector<std::vector<fptype>> ff){

    size_t nEntries = signalDalitz->getCachedWave(0).size();
    size_t n_res = signalDalitz->getDecayInfo().resonances.size();
    fptype sum = 0;

    std::cout << "nEntries= " << nEntries << '\n';
    for(size_t i = 0; i < n_res ; i++){

        for(size_t j = 0; j< n_res ; j++){
            std::cout << "FF[" << i << "," << j <<"]= " << ff[i][j] << std::endl;

        }

        sum+=ff[i][i];
    }

    std::cout << "Sum[i,i]= " << sum << std::endl;
}

void saveParameters(const std::vector<ROOT::Minuit2::MinuitParameter> &param, fptype fcn, size_t nbins, fptype totalPdf){

    std::vector<fptype> v;

    for(size_t i = 0 ; i < param.size() ; i++){

        if(param[i].IsConst() || param[i].IsFixed()){

            continue;

        }else{

            v.push_back(param[i].Value());
            v.push_back(param[i].Error());

        }

    }



    v.push_back(fcn);
    v.push_back(totalPdf);
    v.push_back(nbins);

    std::ofstream output_file("fitResults.txt",std::ofstream::out | std::ofstream::app);
    std::ostream_iterator<std::string> output_iterator(output_file, "\t");
    std::transform(v.begin(), v.end(), output_iterator,
                   [](const fptype &v){return std::to_string(v);});

    }

void runToyFit(std::string toyFileName,size_t nbins) {

    m12.setNumBins(nbins);
    m13.setNumBins(nbins);
    getToyData(toyFileName,true);

    GOOFIT_INFO("Number of events in dataset: {}", data->getNumEvents());

    if(signalDalitz == nullptr){
        signalDalitz = makeSignalPdf();
    }
    comps.clear();
    comps.push_back(signalDalitz);
    ProdPdf *overallSignal = new ProdPdf("overallSignal", comps);
    overallSignal->setData(data);
    signalDalitz->setDataSize(data->getNumEvents());

    signalDalitz->copyParams();
    GOOFIT_INFO("Initial Normalization Value: {}", signalDalitz->normalize());

    FitManagerMinuit2 fitter(overallSignal);
    fitter.setVerbosity(verbosity);

    for(int i = 0; i < HH_bin_limits.size(); i++) {
        pwa_coefs_amp[i].setFixed(false);
        pwa_coefs_phs[i].setFixed(false);
    }

    auto func_min = fitter.fit(); //Minimizer
    makeToyDalitzPdfPlots(overallSignal);

    GOOFIT_INFO("Final Normalization Value: {}", signalDalitz->normalize());

    auto ff = signalDalitz->fit_fractions();

    PrintFF(ff);
}

void FitStudies(std::string toyFileName,size_t nbins) {

    m12.setNumBins(nbins);
    m13.setNumBins(nbins);
    getToyData(toyFileName,false);

    if(signalDalitz == nullptr){
        signalDalitz = makeSignalPdf();
    }

    comps.clear();
    comps.push_back(signalDalitz);
    ProdPdf *overallSignal = new ProdPdf("overallSignal", comps);
    overallSignal->setData(data);
    signalDalitz->setDataSize(data->getNumEvents());

    FitManagerMinuit2 fitter(overallSignal);
    fitter.setVerbosity(0);

    for(int i = 0; i < HH_bin_limits.size(); i++) {
        pwa_coefs_amp[i].setFixed(false);
        pwa_coefs_phs[i].setFixed(false);
    }

    auto func_min = fitter.fit(); //Minimizer

    auto param = fitter.getParams()->Parameters();

    saveParameters(param,func_min.Fval(),nbins,signalDalitz->normalize());
}



void normStudies(std::string toyFileName, size_t begin, size_t end , size_t step){

    GOOFIT_INFO("Start !");

do{

    FitStudies(toyFileName,begin);
    begin+=step;

}while(begin != end);

    GOOFIT_INFO("End !");
}


int main(int argc, char **argv) {

    int sample_number = 0;

    GooFit::Application app{"D2K3_toy", argc, argv};
    app.add_option("-v,--verbose", verbosity, "Set the verbosity (to 0 for example", true);
    app.add_option("-i,--int", sample_number, "sample number", true)->required();

    size_t begin, end, step = 0;
    auto ns = app.add_subcommand("st");
    ns->add_option("-b",begin,"initial value");
    ns->add_option("-e",end,"initial value");
    ns->add_option("-s",step,"initial value");


    size_t nbins = 1000;
    auto fit = app.add_subcommand("fit");
    fit->add_option("-n",nbins,"number of bins")->required();


    int sample_size, n_integrations = 0;
    auto run = app.add_subcommand("run");
    run->add_option("-S", sample_size, "sample size")->required();
	run->add_option("-N", n_integrations, "Number of integrations")->required();

    size_t events = 100000;
    auto gen = app.add_subcommand("gen");
    gen->add_option("-e,--events", events, "The number of events to generate", true);

    /// Must get 1 or more subcommands
    app.require_subcommand();

    GOOFIT_PARSE(app);

    std::string name = fmt::format("dalitz_mytoyMC_{0:03}.txt",sample_number);

    /// Make the plot directory if it does not exist
    std::string command = "mkdir -p plots";
    if(system(command.c_str()) != 0)
        throw GooFit::GeneralError("Making `plots` directory failed");

    GooFit::setROOTStyle();


    if(*ns) {
        CLI::AutoTimer timer("Studies");
        normStudies(name, begin, end, step);
        std::cout << "\n\n";
    }
    if(*gen){
        CLI::AutoTimer timer("MC generation");
        runToyGeneration(name, events);
    }

    if(*fit) {
        CLI::AutoTimer timer("Fit");
        runToyFit(name, nbins);
    }
    if(*run) {
        CLI::AutoTimer timer("Integration");
        runIntegration(sample_size,n_integrations);
        std::cout << "\n\n";
    }


    return 0;
}
