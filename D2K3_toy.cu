#include "hip/hip_runtime.h"
// ROOT stuff
#include <TRandom.h>
#include <TCanvas.h> 
#include <TFile.h> 
#include <TH1F.h> 
#include <TH2F.h>
#include <TStyle.h> 
#include <TRandom3.h> 
#include <TLegend.h> 
#include <TText.h> 
#include <TLine.h> 
#include <TMath.h>
// System stuff
#include <fstream> 
#include <sys/time.h>
#include <sys/times.h>

// GooFit stuff
#include <goofit/Variable.h> 
#include <goofit/PDFs/basic/PolynomialPdf.h> 
#include <goofit/PDFs/physics/DalitzPlotPdf.h> 
#include <goofit/PDFs/physics/DalitzVetoPdf.h> 
#include <goofit/PDFs/physics/ResonancePdf.h> 
#include <goofit/PDFs/combine/AddPdf.h>
#include <goofit/PDFs/combine/ProdPdf.h>
#include <goofit/PDFs/GooPdf.h> 
#include <goofit/FitManager.h> 
#include <goofit/UnbinnedDataSet.h>
#include <goofit/Application.h>

using namespace std;
using namespace GooFit;

TCanvas* foo; 
TCanvas* foodal; 
timeval startTime, stopTime, totalTime;
clock_t startCPU, stopCPU; 
tms startProc, stopProc; 
UnbinnedDataSet* data = 0; 

Observable m12("m12",0.0,3.0);
Observable m13("m13",0.0,3.0);

EventNumber eventNumber("eventNumber");
bool fitMasses = false; 
Variable fixedRhoMass("rho_mass", 0.7758, 0.01, 0.7, 0.8);
Variable fixedRhoWidth("rho_width", 0.1503, 0.01, 0.1, 0.2); 

const fptype _mD0 = 1.86484; 
const fptype piPlusMass = 0.13957018;
const fptype piZeroMass = 0.1349766;
const fptype D1Mass = piZeroMass;
const fptype D2Mass = piPlusMass;
const fptype D3Mass = piPlusMass;
const fptype D1Mass2 = D1Mass*D1Mass;
const fptype D2Mass2 = D2Mass*D2Mass;
const fptype D3Mass2 = D3Mass*D3Mass;
const fptype MMass = _mD0;
const fptype MMass2 = MMass*MMass;
const fptype MMass2inv = 1./MMass2; 

// Constants used in more than one PDF component. 
Variable  motherM("motherM", MMass);
Variable dau1M("dau1M", D1Mass);
Variable dau2M("dau2M", D2Mass);
Variable dau3M("dau3M", D3Mass);
Variable massSum("massSum", MMass2 + D1Mass2+D2Mass2+D3Mass2); // = 3.53481 
Variable constantOne("constantOne", 1); 
Variable constantZero("constantZero", 0); 

std::vector<PdfBase*> comps;

// I don't like Globals! Henry
int verbosity = 3;

GooPdf* kzero_veto = 0; 
char strbuffer[1000]; 
double mesonRad  = 1.5;
DalitzPlotPdf* signalDalitz; 

void makeToyDalitzData (GooPdf* overallSignal, const int iSeed = 0, string datadir = ".", const int nTotal = 1e5 ) ;

DalitzPlotPdf* makeSignalPdf (GooPdf* eff = 0) ;

fptype cpuGetM23 (fptype massPZ, fptype massPM) {
	return (massSum.getValue() - massPZ - massPM); 
}

bool cpuDalitz (fptype m_12, fptype m_13, fptype bigM = MMass, fptype dm1 = D1Mass, fptype dm2 = D2Mass, fptype dm3 = D3Mass) {
	if (m_12 < pow(dm1 + dm2, 2)) return false; // This m_12 cannot exist, it's less than the square of the (1,2) particle mass.
	if (m_12 > pow(bigM - dm3, 2)) return false;   // This doesn't work either, there's no room for an at-rest 3 daughter. 

	// Calculate energies of 1 and 3 particles in m_12 rest frame. 
	fptype e1star = 0.5 * (m_12 - dm2*dm2 + dm1*dm1) / sqrt(m_12); 
	fptype e3star = 0.5 * (bigM*bigM - m_12 - dm3*dm3) / sqrt(m_12); 

	// Bounds for m_13 at this value of m_12.
	fptype minimum = pow(e1star + e3star, 2) - pow(sqrt(e1star*e1star - dm1*dm1) + sqrt(e3star*e3star - dm3*dm3), 2);
	if (m_13 < minimum) return false;
	fptype maximum = pow(e1star + e3star, 2) - pow(sqrt(e1star*e1star - dm1*dm1) - sqrt(e3star*e3star - dm3*dm3), 2);
	if (m_13 > maximum) return false;

	return true; 
}

void makeToyDalitzData (GooPdf* overallSignal, const int iSeed, string datadir, const int nTotal ) {
	std::vector<Observable> vars;
	vars.push_back(m12);
	vars.push_back(m13);
	vars.push_back(eventNumber);
	data = new UnbinnedDataSet(vars);
	UnbinnedDataSet currData(vars); 
	std::vector<std::vector<double>> pdfValues;
	int ncount = 0;
	TRandom3 donram(iSeed); 
	for (int i = 0; i < (m12.getNumBins()) ; ++i) {
		m12.setValue( m12.getLowerLimit() + (m12.getUpperLimit() - m12.getLowerLimit())*(i + 0.5) / m12.getNumBins() ); 
		for (int j = 0; j < m13.getNumBins(); ++j) {
			m13.setValue(m13.getLowerLimit() + (m13.getUpperLimit() - m13.getLowerLimit())*(j + 0.5) / m13.getNumBins()); 
			if (!cpuDalitz(m12.getValue(), m13.getValue(), _mD0, piZeroMass, piPlusMass, piPlusMass)) continue;
			eventNumber.setValue(ncount);
			ncount++;
			currData.addEvent(); 
		}
	}
	signalDalitz->setDataSize(currData.getNumEvents());
	overallSignal->setData(&currData);
	
	pdfValues = overallSignal->getCompProbsAtDataPoints();
	TH2F dalitzpp0_dat_hist("dalitzpp0_dat_hist", "", m12.getNumBins(), m12.getLowerLimit(), m12.getUpperLimit(), m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());
	dalitzpp0_dat_hist.GetXaxis()->SetTitle("m^{2}(#pi^{+}#pi^{0}) [GeV^{2}]");
	dalitzpp0_dat_hist.GetYaxis()->SetTitle("m^{2}(#pi^{-} #pi^{0}) [GeV^{2}]");
	ncount = 0;
	ofstream writer;
	sprintf(strbuffer, "%s/dalitz_mytoyMC_%03d.txt", datadir.c_str(), iSeed);
	writer.open(strbuffer);
	vector<double> fIntegral;
	fIntegral.push_back(pdfValues[0][0]);
	Int_t ncells = pdfValues[0].size();
	for (unsigned int j = 1; j < ncells; ++j) {
		fIntegral.push_back(pdfValues[0][j]+fIntegral[j-1]);
	}
	for (unsigned int j = 0; j < ncells; ++j)  fIntegral[j] /= fIntegral[ncells-1];
	ncount = 0;
	int nEvents = donram.Poisson(nTotal);
	for (int iEvt = 0;iEvt<nEvents;iEvt++){
		double r = donram.Rndm();
		//Binary search for fIntegral[cell-1] < r < fIntegral[cell]
		int lo = 0, hi = ncells-1, mid = 0;
		while(lo <= hi){
			mid = lo + (hi-lo)/2;
			if( r<=fIntegral[mid]&&(mid==0||r>fIntegral[mid-1])) break;
			else if (r > fIntegral[mid] ) lo = mid+1;
			else hi = mid-1;
		}
		int j = mid;
		double currm12 = currData.getValue(m12, j);
		currm12 += (m12.getUpperLimit() - m12.getLowerLimit())*(donram.Rndm() - 0.5) / m12.getNumBins();
		double currm13 = currData.getValue(m13, j);
		currm13 += (m13.getUpperLimit() - m13.getLowerLimit())*(donram.Rndm() - 0.5) / m13.getNumBins();
		eventNumber.setValue(ncount++);
		dalitzpp0_dat_hist.Fill(currm12, currm13);
		data->addEvent();
		writer << ncount-1 << '\t'<<currm12 << '\t'<<currm13<<std::endl;
	}
	writer.close(); 
	std::cout<<"Entries generated: "<<data->getNumEvents()<<std::endl;
	foodal->cd(); 
	foodal->SetLogz(false);
	dalitzpp0_dat_hist.Draw("colz");
	foodal->SaveAs("dalitzpp0_dat_temp.png");
}

void runToyGeneration(int numFile = 0){
	m12   = Observable("m12",   0.0, 3.0);
	m12.setNumBins(1500);
	//  m12   = Variable("m12",   0.4, 3.0);
	m13   = Observable("m13",   0.0, 3.0); 
	m13.setNumBins(1500);
	eventNumber = EventNumber("eventNumber", 0, INT_MAX);
	signalDalitz = makeSignalPdf(); 
	vector<PdfBase*> comps;
	comps.clear(); 
	comps.push_back(signalDalitz);
	//  comps.push_back(sig0_jsugg); 
	std::cout << "Creating overall PDF\n"; 
	ProdPdf* overallSignal = new ProdPdf("overallSignal", comps);
	gettimeofday(&startTime, NULL);
	startCPU = times(&startProc);
	//  makeToyDalitzData (signalDalitz);
	makeToyDalitzData (overallSignal, numFile);
	stopCPU = times(&stopProc);
	gettimeofday(&stopTime, NULL);
}

void getToyData (std::string toyFileName) {
	TH2F dalitzplot("dalitzplot", "", m12.getNumBins(), m12.getLowerLimit(), m12.getUpperLimit(), m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit()); 
	std::vector<Observable> vars;
	vars.push_back(m12);
	vars.push_back(m13);
	vars.push_back(eventNumber); 
	data = new UnbinnedDataSet(vars); 

	std::ifstream reader;
	reader.open(toyFileName.c_str()); 
	assert(reader.good());
	std::string buffer;
	double dummy = 0; 
	reader >> buffer;
	bool oldtype = false;
	if (buffer == "====") oldtype = true;
	else reader.seekg(0);//Set to the beginning
	while (!reader.eof()) {
		reader >> dummy;
		if (oldtype) reader >> dummy;
		reader >> m12; 
		reader >> m13;
		if (oldtype) { for (int i=0;i<16;i++) reader >> dummy; }

		eventNumber.setValue(data->getNumEvents()); 
		data->addEvent(); 

		dalitzplot.Fill(m12.getValue(), m13.getValue()); 
	}
	reader.close();

	dalitzplot.SetStats(false); 
	dalitzplot.Draw("colz");
	foodal->SaveAs("dalitzplot.png"); 
}

GooPdf* makeKzeroVeto () {
	if (kzero_veto) return kzero_veto; 

	
	Variable minimum("veto_min",0.475*0.475);
	Variable maximum("veto_max", 0.505*0.505);
	VetoInfo kVetoInfo(minimum,maximum,PAIR_23);	

	/*VetoInfo* kVetoInfo = new VetoInfo();	
	kVetoInfo->cyclic_index = PAIR_23; 
	kVetoInfo->minimum = Variable("veto_min", 0.475*0.475);
	kVetoInfo->maximum = Variable("veto_max", 0.505*0.505);*/

	vector<VetoInfo> vetos; vetos.push_back(kVetoInfo); 
        kzero_veto = new DalitzVetoPdf("kzero_veto", m12, m13, motherM, dau1M, dau2M, dau3M, vetos); 

	
	return kzero_veto;
}

DalitzPlotPdf* makeSignalPdf (GooPdf* eff /* Some on/off switch or similar */) {
	DecayInfo3 dtop0pp;
	dtop0pp.motherMass  = MMass; 
	dtop0pp.daug1Mass  = D1Mass;
	dtop0pp.daug2Mass  = D2Mass;
	dtop0pp.daug3Mass  = D3Mass;
	dtop0pp.meson_radius  = 1.5; 


    // Make a random number generater heres

	auto rhop  = new Resonances::RBW("rhop",
			Variable("rhop_amp_real", 1),
			Variable("rhop_amp_imag", 0),
			fixedRhoMass,
			fixedRhoWidth,
			1,
			PAIR_12);


	bool fixAmps = false;

    //auto var_func = [&rand_gen](std::string name, double start, double err) -> Variable {
    //   return fixamps ?
    //       Variable(name, start) : 
    //       Variable(name, start + rand_gen(), err, 0, 0);
    //};

	ResonancePdf* rhom  = new Resonances::RBW("rhom", 
			fixAmps ? Variable("rhom_amp_real", 0.714) : 
			Variable("rhom_amp_real",  0.714 /* + rand_gen() */, 0.001, 0, 0),
			fixAmps ? Variable("rhom_amp_imag", -0.025) :
			Variable("rhom_amp_imag", -0.025, 0.1, 0, 0),
			fixedRhoMass,
			fixedRhoWidth,
			1,
			PAIR_13);

	ResonancePdf* rho0  = new Resonances::RBW("rho0", 
			fixAmps ? Variable("rho0_amp_real", 0.565) : 
			Variable("rho0_amp_real", 0.565, 0.001, 0, 0),
			fixAmps ? Variable("rho0_amp_imag", 0.164) :
			Variable("rho0_amp_imag", 0.164, 0.1, 0, 0),
			fixedRhoMass,
			fixedRhoWidth,
			1,
			PAIR_23);

	Variable sharedMass("rhop_1450_mass", 1.465, 0.01, 1.0, 2.0);
	Variable shareWidth("rhop_1450_width", 0.400, 0.01, 0.01, 5.0); 

	ResonancePdf* rhop_1450  = new Resonances::RBW("rhop_1450", 
			fixAmps ? Variable("rhop_1450_amp_real", -0.174) : 
			Variable("rhop_1450_amp_real", -0.174, 0.001, 0, 0),
			fixAmps ? Variable("rhop_1450_amp_imag", -0.117) :
			Variable("rhop_1450_amp_imag", -0.117, 0.1, 0, 0),
			sharedMass,
			shareWidth,
			1,
			PAIR_12);

	ResonancePdf* rho0_1450  = new Resonances::RBW("rho0_1450", 
			fixAmps ? Variable("rho0_1450_amp_real", 0.325) : 
			Variable("rho0_1450_amp_real", 0.325, 0.001, 0, 0),
			fixAmps ? Variable("rho0_1450_amp_imag", 0.057) : 
			Variable("rho0_1450_amp_imag", 0.057, 0.1, 0, 0),  
			sharedMass,
			shareWidth,
			1,
			PAIR_23);

	ResonancePdf* rhom_1450  = new Resonances::RBW("rhom_1450", 
			fixAmps ? Variable("rhom_1450_amp_real", 0.788) : 
			Variable("rhom_1450_amp_real", 0.788, 0.001, 0, 0),
			fixAmps ? Variable("rhom_1450_amp_imag", 0.226) : 
			Variable("rhom_1450_amp_imag", 0.226, 0.1, 0, 0),  
			sharedMass,
			shareWidth,
			1,
			PAIR_13);

	sharedMass = Variable("rhop_1700_mass",  1.720, 0.01, 1.6, 1.9);
	shareWidth = Variable("rhop_1700_width", 0.250, 0.01, 0.1, 1.0); 


	ResonancePdf* rhop_1700  = new Resonances::RBW("rhop_1700", 
			fixAmps ? Variable("rhop_1700_amp_real", 2.151) : 
			Variable("rhop_1700_amp_real",  2.151, 0.001, 0, 0),
			fixAmps ? Variable("rhop_1700_amp_imag", -0.658) : 
			Variable("rhop_1700_amp_imag", -0.658, 0.1, 0, 0),  
			sharedMass,
			shareWidth,
			1,
			PAIR_12);

	ResonancePdf* rho0_1700  = new Resonances::RBW("rho0_1700", 
			fixAmps ? Variable("rho0_1700_amp_real",  2.400) : 
			Variable("rho0_1700_amp_real",  2.400, 0.001, 0, 0),
			fixAmps ? Variable("rho0_1700_amp_imag", -0.734) : 
			Variable("rho0_1700_amp_imag", -0.734, 0.1, 0, 0),  
			sharedMass,
			shareWidth,
			1,
			PAIR_23);

	ResonancePdf* rhom_1700  = new Resonances::RBW("rhom_1700", 
			fixAmps ? Variable("rhom_1700_amp_real",  1.286) : 
			Variable("rhom_1700_amp_real",  1.286, 0.001, 0, 0),
			fixAmps ? Variable("rhom_1700_amp_imag", -1.532) : 
			Variable("rhom_1700_amp_imag", -1.532, 0.1, 0, 0),  
			sharedMass,
			shareWidth,
			1,
			PAIR_13);

	auto f0_980  = new Resonances::FLATTE("f0_980", 
			fixAmps ? Variable("f0_980_amp_real",  0.008 * (-MMass2)) : Variable("f0_980_amp_real",  0.008 * (-MMass2), 0.001, 0, 0),
			fixAmps ? Variable("f0_980_amp_imag", -0.013 * (-MMass2)) : Variable("f0_980_amp_imag", -0.013 * (-MMass2), 0.1, 0, 0),  
			Variable("f0_980_mass",     0.9399/*0.980*/, 0.01, 0.8, 1.2),
			Variable("f0_980_width",    0.199/*0.044*/, 0.001, 0.001, 0.08),
			Variable("f0_980_rg2og1",    3.0, 0.1, 1e-3, 10),
			PAIR_23, false);

	ResonancePdf* f0_1370  = new Resonances::RBW("f0_1370", 
			fixAmps ? Variable("f0_1370_amp_real", -0.058 * (-MMass2)) : 
			Variable("f0_1370_amp_real", -0.058 * (-MMass2), 0.001, 0, 0),
			fixAmps ? Variable("f0_1370_amp_imag",  0.026 * (-MMass2)) : 
			Variable("f0_1370_amp_imag",  0.026 * (-MMass2), 0.1, 0, 0),  
			Variable("f0_1370_mass",     1.434, 0.01, 1.2, 1.6),
			Variable("f0_1370_width",    0.173, 0.01, 0.01, 0.4),
			(unsigned int)0,
			PAIR_23);

	ResonancePdf* f0_1500  = new Resonances::RBW("f0_1500", 
			fixAmps ? Variable("f0_1500_amp_real", 0.057 * (-MMass2)) : 
			Variable("f0_1500_amp_real", 0.057 * (-MMass2), 0.001, 0, 0),
			fixAmps ? Variable("f0_1500_amp_imag", 0.012 * (-MMass2)) : 
			Variable("f0_1500_amp_imag", 0.012 * (-MMass2), 0.1, 0, 0),  
			Variable("f0_1500_mass",     1.507, 0.01, 1.3, 1.7),
			Variable("f0_1500_width",    0.109, 0.01, 0.01, 0.3),
			(unsigned int)0,
			PAIR_23);

	ResonancePdf* f0_1710  = new Resonances::RBW("f0_1710", 
			fixAmps ? Variable("f0_1710_amp_real", 0.070 * (-MMass2)) : 
			Variable("f0_1710_amp_real", 0.070 * (-MMass2), 0.001, 0, 0),
			fixAmps ? Variable("f0_1710_amp_imag", 0.087 * (-MMass2)) : 
			Variable("f0_1710_amp_imag", 0.087 * (-MMass2), 0.1, 0, 0),  
			Variable("f0_1710_mass",     1.714, 0.01, 1.5, 2.9), 
			Variable("f0_1710_width",    0.140, 0.01, 0.01, 0.5),
			(unsigned int)0,
			PAIR_23);

	ResonancePdf* f2_1270  = new Resonances::RBW("f2_1270", 
			fixAmps ? Variable("f2_1270_amp_real", -1.027 * (-MMass2inv)) : 
			Variable("f2_1270_amp_real", -1.027 * (-MMass2inv), 0.001, 0, 0),
			fixAmps ? Variable("f2_1270_amp_imag", -0.162 * (-MMass2inv)) : 
			Variable("f2_1270_amp_imag", -0.162 * (-MMass2inv), 0.1, 0, 0),  
			Variable("f2_1270_mass",     1.2754, 0.01, 1.0, 1.5),
			Variable("f2_1270_width",    0.1851, 0.01, 0.01, 0.4),
			2,
			PAIR_23);

	ResonancePdf* f0_600  = new Resonances::RBW("f0_600", 
			fixAmps ? Variable("f0_600_amp_real", 0.068 * (-MMass2)) : 
			Variable("f0_600_amp_real", 0.068 * (-MMass2), 0.001, 0, 0),
			fixAmps ? Variable("f0_600_amp_imag", 0.010 * (-MMass2)) : 
			Variable("f0_600_amp_imag", 0.010 * (-MMass2), 0.1, 0, 0),  
			Variable("f0_600_mass",     0.500, 0.01, 0.3, 0.7),
			Variable("f0_600_width",    0.400, 0.01, 0.2, 0.6), 
			(unsigned int)0,
			PAIR_23);

	ResonancePdf* nonr  = new Resonances::NonRes("nonr",
			fixAmps ? Variable("nonr_amp_real", 0.5595 * (-1)) : 
			Variable("nonr_amp_real", 0.5595 * (-1),   0.001, 0, 0),
			fixAmps ? Variable("nonr_amp_imag", -0.108761 * (-1)) : 
			Variable("nonr_amp_imag", -0.108761* (-1), 0.1, 0, 0)); 

	dtop0pp.resonances.push_back(nonr); 
	dtop0pp.resonances.push_back(rhop);
	dtop0pp.resonances.push_back(rho0); 
	dtop0pp.resonances.push_back(rhom); 
	dtop0pp.resonances.push_back(rhop_1450); 
	dtop0pp.resonances.push_back(rho0_1450); 
	dtop0pp.resonances.push_back(rhom_1450); 
	dtop0pp.resonances.push_back(rhop_1700); 
	dtop0pp.resonances.push_back(rho0_1700); 
	dtop0pp.resonances.push_back(rhom_1700); 
	dtop0pp.resonances.push_back(f0_980); 
	dtop0pp.resonances.push_back(f0_1370); 
	dtop0pp.resonances.push_back(f0_1500); 
	dtop0pp.resonances.push_back(f0_1710); 
	dtop0pp.resonances.push_back(f2_1270); 
	dtop0pp.resonances.push_back(f0_600); 

	if (!fitMasses) {
		for (vector<ResonancePdf*>::iterator res = dtop0pp.resonances.begin(); res != dtop0pp.resonances.end(); ++res) {
			(*res)->setParameterConstantness(true); 
		}
	}

	if (!eff) {
		// By default create a constant efficiency. 
		vector<Variable> offsets;
		vector<Observable> observables;
		vector<Variable> coefficients; 

		observables.push_back(m12);
		observables.push_back(m13);
		offsets.push_back(constantZero);
		offsets.push_back(constantZero);
		coefficients.push_back(constantOne); 
		eff = new PolynomialPdf("constantEff", observables, coefficients, offsets, 0);
	}
	comps.clear();
	comps.push_back(eff);
	if (!kzero_veto) makeKzeroVeto();
	comps.push_back(kzero_veto);
	ProdPdf* effWithVeto = new ProdPdf("effWithVeto", comps);

	return new DalitzPlotPdf("signalPDF", m12, m13, eventNumber, dtop0pp, effWithVeto);
}

void drawFitPlotsWithPulls(TH1* hd, TH1* ht, string plotdir){
	const char* hname = hd->GetName();
	char obsname[10];
	for (int i=0;;i++) {
		if (hname[i]=='_') obsname[i] = '\0';
		else obsname[i] = hname[i];
		if (obsname[i] == '\0') break;
	}
	ht->Scale(hd->Integral()/ht->Integral());
	foo->cd(); 
	foo->Clear();
	ht->Draw("l");
	hd->Draw("epsame");
	sprintf(strbuffer, "%s/%s_fit.png", plotdir.c_str(), obsname);
	foo->SaveAs(strbuffer);
	sprintf(strbuffer, "%s/%s_fit.pdf", plotdir.c_str(), obsname);
	foo->SaveAs(strbuffer);
	/*    sprintf(strbuffer, "%s/%s_fit_log.pdf", plotdir.c_str(), obsname);
		  foo->SaveAs(strbuffer);*/
}

void makeToyDalitzPdfPlots (GooPdf* overallSignal, string plotdir = "plots") {
	TH1F m12_dat_hist("m12_dat_hist", "", m12.getNumBins(), m12.getLowerLimit(), m12.getUpperLimit());
	m12_dat_hist.SetStats(false); 
	m12_dat_hist.SetMarkerStyle(8); 
	m12_dat_hist.SetMarkerSize(1.2);
	m12_dat_hist.GetXaxis()->SetTitle("m^{2}(#pi^{+} #pi^{0}) [GeV]");
	sprintf(strbuffer, "Events / %.1f MeV", 1e3*m12_dat_hist.GetBinWidth(1));
	m12_dat_hist.GetYaxis()->SetTitle(strbuffer); 
	TH1F m12_pdf_hist("m12_pdf_hist", "", m12.getNumBins(), m12.getLowerLimit(), m12.getUpperLimit());
	m12_pdf_hist.SetStats(false); 
	m12_pdf_hist.SetLineColor(kBlue); 
	m12_pdf_hist.SetLineWidth(3); 
	TH1F m13_dat_hist("m13_dat_hist", "", m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());
	m13_dat_hist.SetStats(false); 
	m13_dat_hist.SetMarkerStyle(8); 
	m13_dat_hist.SetMarkerSize(1.2);
	m13_dat_hist.GetXaxis()->SetTitle("m^{2}(#pi^{-} #pi^{0}) [GeV]");
	sprintf(strbuffer, "Events / %.1f MeV", 1e3*m13_dat_hist.GetBinWidth(1));
	m13_dat_hist.GetYaxis()->SetTitle(strbuffer); 
	TH1F m13_pdf_hist("m13_pdf_hist", "", m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());
	m13_pdf_hist.SetStats(false); 
	m13_pdf_hist.SetLineColor(kBlue); 
	m13_pdf_hist.SetLineWidth(3); 
	TH1F m23_dat_hist("m23_dat_hist", "", m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());
	m23_dat_hist.SetStats(false); 
	m23_dat_hist.SetMarkerStyle(8); 
	m23_dat_hist.SetMarkerSize(1.2);
	m23_dat_hist.GetXaxis()->SetTitle("m^{2}(#pi^{+} #pi^{-}) [GeV]");
	sprintf(strbuffer, "Events / %.1f MeV", 1e3*m13_dat_hist.GetBinWidth(1));
	m23_dat_hist.GetYaxis()->SetTitle(strbuffer); 
	TH1F m23_pdf_hist("m23_pdf_hist", "", m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());
	m23_pdf_hist.SetStats(false); 
	m23_pdf_hist.SetLineColor(kBlue); 
	m23_pdf_hist.SetLineWidth(3); 
	double totalPdf = 0; 
	double totalDat = 0; 
	TH2F dalitzpp0_dat_hist("dalitzpp0_dat_hist", "", m12.getNumBins(), m12.getLowerLimit(), m12.getUpperLimit(), m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());
	dalitzpp0_dat_hist.SetStats(false); 
	dalitzpp0_dat_hist.GetXaxis()->SetTitle("m^{2}(#pi^{+} #pi^{0}) [GeV]");
	dalitzpp0_dat_hist.GetYaxis()->SetTitle("m^{2}(#pi^{-} #pi^{0}) [GeV]");
	TH2F dalitzpp0_pdf_hist("dalitzpp0_pdf_hist", "", m12.getNumBins(), m12.getLowerLimit(), m12.getUpperLimit(), m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());
	/*  dalitzpp0_pdf_hist.GetXaxis()->SetTitle("m^{2}(K^{-} #pi^{0}) [GeV^{2}]");
		dalitzpp0_pdf_hist.GetYaxis()->SetTitle("m^{2}(K^{-} #pi^{+}) [GeV^{2}]");*/
	dalitzpp0_pdf_hist.GetXaxis()->SetTitle("m^{2}(#pi^{+} #pi^{0}) [GeV^{2}]");
	dalitzpp0_pdf_hist.GetYaxis()->SetTitle("m^{2}(#pi^{-} #pi^{0}) [GeV^{2}]");
	dalitzpp0_pdf_hist.SetStats(false); 
	std::vector<Observable> vars;
	vars.push_back(m12);
	vars.push_back(m13);
	vars.push_back(eventNumber); 
	UnbinnedDataSet currData(vars); 
	int evtCounter = 0; 

	for (int i = 0; i < m12.getNumBins(); ++i) {
		m12.setValue(m12.getLowerLimit() + (m12.getUpperLimit() - m12.getLowerLimit())*(i + 0.5) / m12.getNumBins()); 
		for (int j = 0; j < m13.getNumBins(); ++j) {
			m13.setValue(m13.getLowerLimit() + (m13.getUpperLimit() - m13.getLowerLimit())*(j + 0.5) / m13.getNumBins()); 
			if (!cpuDalitz(m12.getValue(), m13.getValue(), _mD0, piZeroMass, piPlusMass, piPlusMass)) continue;
			eventNumber.setValue(evtCounter); 
			evtCounter++;
			currData.addEvent(); 
		}
	}
	overallSignal->setData(&currData);
	signalDalitz->setDataSize(currData.getNumEvents()); 
	std::vector<std::vector<double> > pdfValues;
	pdfValues = overallSignal->getCompProbsAtDataPoints();
	for (unsigned int j = 0; j < pdfValues[0].size(); ++j) {
		double currm12 = currData.getValue(m12, j);
		double currm13 = currData.getValue(m13, j);

		dalitzpp0_pdf_hist.Fill(currm12, currm13, pdfValues[0][j]);
		m12_pdf_hist.Fill(currm12, pdfValues[0][j]);
		m13_pdf_hist.Fill(currm13, pdfValues[0][j]);
		m23_pdf_hist.Fill(cpuGetM23(currm12, currm13), pdfValues[0][j]); 
		totalPdf     += pdfValues[0][j]; 
	}
	foodal->cd(); 
	foodal->SetLogz(false);
	dalitzpp0_pdf_hist.Draw("colz");
    std::string command = "mkdir -p " + plotdir;
    if (system(command.c_str()) != 0)
        throw GooFit::GeneralError("Making plot directory {} failed", plotdir);
	foodal->SaveAs((plotdir + "/dalitzpp0_pdf.png").c_str());
	/*  m12_pdf_hist.Draw("");
		foodal->SaveAs((plotdir + "/m12_pdf_hist.png").c_str());
		m13_pdf_hist.Draw("");
		foodal->SaveAs((plotdir + "/m13_pdf_hist.png").c_str());
		if (!data) return;*/
	for (unsigned int evt = 0; evt < data->getNumEvents(); ++evt) {
		double data_m12 = data->getValue(m12, evt);
		m12_dat_hist.Fill(data_m12); 
		double data_m13 = data->getValue(m13, evt);
		m13_dat_hist.Fill(data_m13); 
		dalitzpp0_dat_hist.Fill(data_m12, data_m13);
		m23_dat_hist.Fill(cpuGetM23(data_m12, data_m13)); 
		totalDat++; 
	}
	dalitzpp0_dat_hist.Draw("colz");
	foodal->SaveAs((plotdir + "/dalitzpp0_dat.png").c_str());

	drawFitPlotsWithPulls(&m12_dat_hist, &m12_pdf_hist, plotdir);
	drawFitPlotsWithPulls(&m13_dat_hist, &m13_pdf_hist, plotdir);
	drawFitPlotsWithPulls(&m23_dat_hist, &m23_pdf_hist, plotdir);
}

void runToyFit (std::string toyFileName) {
	m12 = Observable("m12", 0, 3);
	m13 = Observable("m13", 0, 3); 
	m12.setNumBins(300);
	m13.setNumBins(300);
	eventNumber = EventNumber("eventNumber", 0, INT_MAX);
	getToyData(toyFileName);

	// EXERCISE 1 (real part): Create a PolynomialPdf which models
	// the efficiency you imposed in the preliminary, and use it in constructing
	// the signal PDF. 

	// EXERCISE 2: Create a K0 veto function and use it as the efficiency. 

	// EXERCISE 3: Make the efficiency a product of the two functions
	// from the previous exercises.

	signalDalitz = makeSignalPdf(); 
	comps.clear();
	comps.push_back(signalDalitz);
	ProdPdf* overallSignal = new ProdPdf("overallSignal", comps);
	overallSignal->setData(data); 
	signalDalitz->setDataSize(data->getNumEvents()); 
	FitManager datapdf(overallSignal); 

	gettimeofday(&startTime, NULL);
	startCPU = times(&startProc);
    datapdf.setVerbosity(verbosity); // Maybe make optional? With a command line switch?
	datapdf.fit(); 
	stopCPU = times(&stopProc);
	gettimeofday(&stopTime, NULL);

	//Get the fractions w/ uncertainties
	//vector<double> fracList;
	//signalDalitz->getFractions(fracList);
	/*  const int nRes = fracList.size();
		vector <float> fractions[nRes];
		float mean[nRes];
		float rms[nRes];
		for (int ii=0;ii<nRes;ii++) mean[ii] = rms[ii] = 0;
		for (int ii=0;ii<nSamples;ii++){
		datapdf.loadSample(ii);
		signalDalitz->getFractions(fracList);
		for (int jj=0;jj<nRes; jj++) {
		fractions[jj].push_back(fracList[jj]);
		mean[jj] += fracList[jj];
		rms[jj] += fracList[jj]*fracList[jj];
		}
		}
		TH1F* hFracs[nRes];
		TFile * froot = new TFile("fractionHists.root", "recreate");
		for (int ii=0;ii<nRes;ii++) {
		mean[ii] /= nSamples;
		rms[ii] = sqrt(rms[ii]/nSamples-mean[ii]*mean[ii]);
		sprintf(strbuffer, "hfrac_res%d", ii);
		hFracs[ii] = new TH1F(strbuffer, "", 100, mean[ii]-4*rms[ii], mean[ii]+4*rms[ii]);
		for (int jj=0;jj<nSamples;jj++)
		hFracs[ii]->Fill(fractions[ii][jj]);
		hFracs[ii]->Write();
		}
		froot->Close();*/

	makeToyDalitzPdfPlots(overallSignal);   
}

int main (int argc, char** argv) {

    GooFit::Application app{"D2K3_toy", argc, argv};
    app.add_option("-v,--verbose", verbosity, "Set the verbosity (to 0 for example", true);

    int fit_value;
    std::string name = "dalitz_mytoyMC_000.txt";

    auto fit = app.add_subcommand("fit");
    fit->add_option("-i,--int", fit_value, "A number to load");
    auto name_opt = fit->add_option("-n,--name,name", name, "The filename to load", true)
        ->excludes("--int");

    int value;
    auto gen = app.add_subcommand("gen");
    gen->add_option("value", value, "The number to generate")
        ->required();
    
    app.require_subcommand(1);

    GOOFIT_PARSE(app);

    if(name_opt->count())
        name = fmt::format("dalitz_mytoyMC_{0:3}.txt", fit_value);

	gStyle->SetCanvasBorderMode(0);
	gStyle->SetCanvasColor(10);
	gStyle->SetFrameFillColor(10);
	gStyle->SetFrameBorderMode(0);
	gStyle->SetPadColor(0);
	gStyle->SetTitleColor(1);
	gStyle->SetStatColor(0);
	gStyle->SetFillColor(0);
	gStyle->SetFuncWidth(1);
	gStyle->SetLineWidth(1);
	gStyle->SetLineColor(1);
	gStyle->SetPalette(1, 0);
	foo = new TCanvas(); 
	foodal = new TCanvas(); 
	foodal->Size(10, 10);


    if(*fit)
	    runToyFit(name);
    if(*gen)
	    runToyGeneration(value); 

	// Print total minimization time
	double myCPU = stopCPU - startCPU;
	double totalCPU = myCPU; 

	timersub(&stopTime, &startTime, &totalTime);
	std::cout << "Wallclock time  : " << totalTime.tv_sec + totalTime.tv_usec/1000000.0 << " seconds." << std::endl;
	std::cout << "CPU time: " << (myCPU / CLOCKS_PER_SEC) << std::endl; 
	std::cout << "Total CPU time: " << (totalCPU / CLOCKS_PER_SEC) << std::endl; 
	myCPU = stopProc.tms_utime - startProc.tms_utime;
	std::cout << "Processor time: " << (myCPU / CLOCKS_PER_SEC) << std::endl;

	return 0; 
}
