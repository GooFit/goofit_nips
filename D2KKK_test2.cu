#include "hip/hip_runtime.h"
// ROOT stuff
#include <TRandom.h>
#include <TCanvas.h>
#include <TFile.h>
#include <TH1F.h>
#include <TH2F.h>
#include <TStyle.h>
#include <TRandom3.h>
#include <TLegend.h>
#include <TText.h>
#include <TLine.h>
#include <TMath.h>
#include <TApplication.h>
#include <TTree.h>

// System stuff
#include <fstream>
#include <sys/time.h>
#include <sys/times.h>
#include <random>
#include <CLI/Timer.hpp>
#include <stdio.h>
#include <iostream>
#include <math.h>

// GooFit stuff
#include <goofit/Variable.h>
#include <goofit/PDFs/basic/PolynomialPdf.h>
#include <goofit/PDFs/physics/DalitzPlotPdf.h>
#include <goofit/PDFs/physics/DalitzVetoPdf.h>
#include <goofit/PDFs/physics/ResonancePdf.h>
#include <goofit/PDFs/combine/AddPdf.h>
#include <goofit/PDFs/combine/ProdPdf.h>
#include <goofit/PDFs/GooPdf.h>
#include <goofit/FitManager.h>
#include <goofit/UnbinnedDataSet.h>
#include <goofit/BinnedDataSet.h>
#include <goofit/Application.h>
#include <goofit/PDFs/basic/SmoothHistogramPdf.h>

using namespace std;
using namespace GooFit;

const int PI = 3.14159265358979323846;

TCanvas* foo;
TCanvas* foodal;
timeval startTime, stopTime, totalTime;
clock_t startCPU, stopCPU;
tms startProc, stopProc;
UnbinnedDataSet* data = 0;
const unsigned int nbins = 1000;
TH2F* weightHistogram = 0;
TH2F* bkgHistogram    = 0;
TH2F* underlyingBins  = 0;

// How many events will be generated for Eff Bkg?
const int NevG = 1e7;

// PWA INPUT FILE NAME
const string  pwa_file = "files/PWA_COEFFS_50.txt";


// FIT OR JUST PLOT?
bool fit = true;

Observable m12("m12", 0.9, 2.0);
Observable m13("m13", 0.9, 2.0);
EventNumber eventNumber("eventNumber");
bool fitMasses = false;
Variable fixedPhiMass("phi_mass", 1.019461, 0.01, 0.7, 1.8);
Variable fixedPhiWidth("phi_width", 0.004266, 0.001, 1e-5, 1e-1);

const fptype _mDp = 1.86962;
const fptype KPlusMass = 0.493677;
double V = (m12.getUpperLimit() - m12.getLowerLimit())*(m13.getUpperLimit() - m13.getLowerLimit()); //Volume


const fptype D1Mass = KPlusMass;//
const fptype D2Mass = KPlusMass;
const fptype D3Mass = KPlusMass;
const fptype D1Mass2 = D1Mass*D1Mass;
const fptype D2Mass2 = D2Mass*D2Mass;
const fptype D3Mass2 = D3Mass*D3Mass;
const fptype MMass = _mDp;
const fptype MMass2 = MMass*MMass;


//const fptype MMass2inv = 1./MMass2;

// Constants used in more than one PDF component.
Variable  motherM("motherM", MMass);
Variable dau1M("dau1M", D1Mass);
Variable dau2M("dau2M", D2Mass);
Variable dau3M("dau3M", D3Mass);
Variable massSum("massSum", MMass2 + D1Mass2+D2Mass2+D3Mass2); // = 3.53481
Variable constantOne("constantOne", 1);
Variable constantZero("constantZero", 0);

std::vector<PdfBase*> comps;

// I don't like Globals! Henry
int verbosity = 3;

GooPdf* kzero_veto = 0;
char strbuffer[1000];
double mesonRad  = 1.5;
DalitzPlotPdf* signalDalitz;
bool doEffSwap = true;
bool saveEffPlot = true;
bool saveBkgPlot = true;

void makeToyDalitzData (GooPdf* overallSignal, const int iSeed = 0, string datadir = ".", const int nTotal = 1.e6 ) ;

DalitzPlotPdf* makeSignalPdf (GooPdf* eff = 0, bool fixAmps = false) ;

fptype cpuGetM23 (fptype massPZ, fptype massPM) {
	return (massSum.getValue() - massPZ - massPM);
}

bool cpuDalitz (fptype m_12, fptype m_13, fptype bigM = MMass, fptype dm1 = D1Mass, fptype dm2 = D2Mass, fptype dm3 = D3Mass) {
	if (m_12 < pow(dm1 + dm2, 2)) return false; // This m_12 cannot exist, it's less than the square of the (1,2) particle mass.
	if (m_12 > pow(bigM - dm3, 2)) return false;   // This doesn't work either, there's no room for an at-rest 3 daughter.

	// Calculate energies of 1 and 3 particles in m_12 rest frame.
	fptype e1star = 0.5 * (m_12 - dm2*dm2 + dm1*dm1) / sqrt(m_12);
	fptype e3star = 0.5 * (bigM*bigM - m_12 - dm3*dm3) / sqrt(m_12);

	// Bounds for m_13 at this value of m_12.
	fptype minimum = pow(e1star + e3star, 2) - pow(sqrt(e1star*e1star - dm1*dm1) + sqrt(e3star*e3star - dm3*dm3), 2);
	if (m_13 < minimum) return false;
	fptype maximum = pow(e1star + e3star, 2) - pow(sqrt(e1star*e1star - dm1*dm1) - sqrt(e3star*e3star - dm3*dm3), 2);
	if (m_13 > maximum) return false;

	return true;
}

void makeToyDalitzData (GooPdf* overallSignal, const int iSeed, string datadir, const int nTotal ) {
	std::vector<Observable> vars;
	vars.push_back(m12);
	vars.push_back(m13);
	vars.push_back(eventNumber);
	data = new UnbinnedDataSet(vars);
	UnbinnedDataSet currData(vars);
	std::vector<std::vector<double>> pdfValues;
	int ncount = 0;
	TRandom3 donram(iSeed);
	for (int i = 0; i < (m12.getNumBins()) ; ++i) {
		m12.setValue( m12.getLowerLimit() + (m12.getUpperLimit() - m12.getLowerLimit())*(i + 0.5) / m12.getNumBins() );
		for (int j = 0; j < m13.getNumBins(); ++j) {
			m13.setValue(m13.getLowerLimit() + (m13.getUpperLimit() - m13.getLowerLimit())*(j + 0.5) / m13.getNumBins());
			if (!cpuDalitz(m12.getValue(), m13.getValue(), MMass , D1Mass, D2Mass,D3Mass)) continue;
			eventNumber.setValue(ncount);
			ncount++;
			currData.addEvent();
		}
	}
	signalDalitz->setDataSize(currData.getNumEvents());
	overallSignal->setData(&currData);

	pdfValues = overallSignal->getCompProbsAtDataPoints();
	TH2F dalitzpp0_dat_hist("dalitzpp0_dat_hist", "", m12.getNumBins(), m12.getLowerLimit(), m12.getUpperLimit(), m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());
	dalitzpp0_dat_hist.GetXaxis()->SetTitle("m^{2}(K^{-}K^{+}) [GeV^{2}]");
	dalitzpp0_dat_hist.GetYaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV^{2}]");
	ncount = 0;
	ofstream writer;
	sprintf(strbuffer, "%s/dalitz_mytoyMC_%03d.txt", datadir.c_str(), iSeed);
	writer.open(strbuffer);
	vector<double> fIntegral;
	fIntegral.push_back(pdfValues[0][0]);
	Int_t ncells = pdfValues[0].size();
	for (unsigned int j = 1; j < ncells; ++j) {
		fIntegral.push_back(pdfValues[0][j]+fIntegral[j-1]);
	}
	for (unsigned int j = 0; j < ncells; ++j)  fIntegral[j] /= fIntegral[ncells-1];
	ncount = 0;
	int nEvents = donram.Poisson(nTotal);
	for (int iEvt = 0;iEvt<nEvents;iEvt++){
		double r = donram.Rndm();
		//Binary search for fIntegral[cell-1] < r < fIntegral[cell]
		int lo = 0, hi = ncells-1, mid = 0;
		while(lo <= hi){
			mid = lo + (hi-lo)/2;
			if( r<=fIntegral[mid]&&(mid==0||r>fIntegral[mid-1])) break;
			else if (r > fIntegral[mid] ) lo = mid+1;
			else hi = mid-1;
		}
		int j = mid;
		double currm12 = currData.getValue(m12, j);
		currm12 += (m12.getUpperLimit() - m12.getLowerLimit())*(donram.Rndm() - 0.5) / m12.getNumBins();
		double currm13 = currData.getValue(m13, j);
		currm13 += (m13.getUpperLimit() - m13.getLowerLimit())*(donram.Rndm() - 0.5) / m13.getNumBins();
		eventNumber.setValue(ncount++);
		dalitzpp0_dat_hist.Fill(currm12, currm13);
		data->addEvent();
		writer << ncount-1 << '\t'<<currm12 << '\t'<<currm13<<std::endl;
	}
	writer.close();
	std::cout<<"Entries generated: "<<data->getNumEvents()<<std::endl;
	foodal->cd();
	foodal->SetLogz(false);
	dalitzpp0_dat_hist.Rebin2D(10,10);
	dalitzpp0_dat_hist.Draw("colz");
	dalitzpp0_dat_hist.SetStats(0);
	//foodal->SaveAs("Dalitz_D2KKK_temp.root");
	foodal->SaveAs("D2KKK_Plots/Dalitz_D2KKK_temp.png");

}

void runToyGeneration(int numFile = 0){
	m12   = Observable("m12",   0.9, 2.0);
	m12.setNumBins(1500);

	m13   = Observable("m13",   0.9, 2.0);
	m13.setNumBins(1500);
	eventNumber = EventNumber("eventNumber", 0, INT_MAX);
	signalDalitz = makeSignalPdf(0,false);
	vector<PdfBase*> comps;
	comps.clear();
	comps.push_back(signalDalitz);

	std::cout << "Creating overall PDF\n";
	ProdPdf* overallSignal = new ProdPdf("overallSignal", comps);
	gettimeofday(&startTime, NULL);
	startCPU = times(&startProc);
	//  makeToyDalitzData (signalDalitz);
	makeToyDalitzData (overallSignal, numFile);
	stopCPU = times(&stopProc);
	gettimeofday(&stopTime, NULL);
}

void getToyData (std::string toyFileName) {
	 TH2F dalitzplot("dalitzplot", "", m12.getNumBins(), m12.getLowerLimit(), m12.getUpperLimit(), m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());
  std::vector<Observable> vars;
  vars.push_back(m12);
  vars.push_back(m13);
  vars.push_back(eventNumber);
  data = new UnbinnedDataSet(vars);
//  const int MAXEVT = 1e4;

  const string suffix = ".root";
  if (toyFileName.rfind(suffix)+suffix.length() == toyFileName.length()){
      std::cout<<"Reading file "<<toyFileName<<std::endl;
      TFile*f = TFile::Open(toyFileName.c_str());
      TTree*t = (TTree*)f->Get("DecayTree");
      //TTree*t = (TTree*)f->Get("newTree");
      std::cout<<"Entries: "<<t->GetEntries()<<std::endl;
      assert(t);
      double m2_12, m2_13;
      //t->SetBranchAddress("s12_KK_DTF", &m2_12);
      //t->SetBranchAddress("s13_KK_DTF", &m2_13);
      t->SetBranchAddress("s12", &m2_12);
      t->SetBranchAddress("s13", &m2_13);
      for (int i=0;i<t->GetEntries()/*&&i<MAXEVT*/;i++){
      //for (int i=0;i<100000;i++){
          t->GetEntry(i);
          m12.setValue(m2_12);
          m13.setValue(m2_13);
          eventNumber.setValue(data->getNumEvents());
          data->addEvent();
          dalitzplot.Fill(m12.getValue(), m13.getValue());
      }
      f->Close();
  }
  else{
  std::ifstream reader;
  reader.open(toyFileName.c_str());
  std::string buffer;
  while (!reader.eof()) {
    reader >> buffer;
    if (buffer == "====") break;
    //std::cout << buffer;
  }

  double dummy = 0;
  while (!reader.eof()) {
    reader >> dummy;
    reader >> dummy;      // m23, m(pi+ pi-), called m12 in processToyRoot convention.
    reader >> m12; // Already swapped according to D* charge. m12 = m(pi+pi0)
    reader >> m13;

    // Errors on Dalitz variables
    reader >> dummy;
    reader >> dummy;
    reader >> dummy;

    reader >> dummy; // Decay time
    reader >> dummy; // sigma_t

    reader >> dummy; // Md0
    reader >> dummy; // deltaM
    reader >> dummy; // ProbSig
    reader >> dummy; // Dst charge
    reader >> dummy; // Run
    reader >> dummy; // Event
    reader >> dummy; // Signal and four bkg fractions.
    reader >> dummy;
    reader >> dummy;
    reader >> dummy;
    reader >> dummy;

    eventNumber.setValue(data->getNumEvents());
    data->addEvent();

    dalitzplot.Fill(m12.getValue(), m13.getValue());
  }}


  dalitzplot.SetStats(0);
  dalitzplot.Draw("colz");
  foodal->SaveAs("dalitzplot_D2KKK_gen.png"); }

/*GooPdf* makeKzeroVeto () {
	if (kzero_veto) return kzero_veto;


	Variable minimum("veto_min",0.475*0.475);
	Variable maximum("veto_max", 0.505*0.505);
	VetoInfo kVetoInfo(minimum,maximum,PAIR_23);

	vector<VetoInfo> vetos; vetos.push_back(kVetoInfo);
        kzero_veto = new DalitzVetoPdf("kzero_veto", m12, m13, motherM, dau1M, dau2M, dau3M, vetos);


	return kzero_veto;
}*/

void createWeightHistogram () {

  TFile*f = TFile::Open("Fit_Input/effspline300.root");
  weightHistogram = (TH2F*)f->Get("eff_spline");
  weightHistogram->SetStats(false);
}

void createBackgroundHistogram () {
  TFile*f = TFile::Open("Fit_Input/bkg_histo_300bins.root");
  bkgHistogram = (TH2F*)f->Get("bkgHist_acc");
  bkgHistogram->SetStats(false);
}

GooPdf* makeEfficiencyPdf () {
  vector<Observable> lvars;
  lvars.push_back(m12);
  lvars.push_back(m13);
  BinnedDataSet* binEffData = new BinnedDataSet(lvars);
  //createWeightHistogram();
  // Now testing your efficiency data by uniformly generating m12,m13 values
  TRandom3 donram(0);
  for (int i = 0; i < NevG; i++){
    do{
    m12.setValue(donram.Uniform(m12.getLowerLimit(), m12.getUpperLimit()));
    m13.setValue(donram.Uniform(m13.getLowerLimit(), m13.getUpperLimit()));
    }while(!cpuDalitz(m12.getValue(), m13.getValue(), MMass , D1Mass, D2Mass,D3Mass));
    //Weight will not be one if the physics boundary crosses the bin square.
    double weight = weightHistogram->GetBinContent(weightHistogram->FindBin(m12.getValue(), m13.getValue()));
    binEffData->addWeightedEvent(weight);
    //if (underlyingBins) underlyingBins->Fill(m12->value, m13->value, weight);
    // Imposing the requirement on efficiency symmetry along m12=m13 line
      if (doEffSwap){
      double swapmass = m12.getValue(); m12.setValue(m13.getValue()); m13.setValue(swapmass);
      weight = weightHistogram->GetBinContent(weightHistogram->FindBin(m12.getValue(), m13.getValue()));
      binEffData->addWeightedEvent(weight);
      //if (underlyingBins) underlyingBins->Fill(m12->value, m13->value, weight);
      //swapmass = m12->value; m12->value = m13->value; m13->value = swapmass;
      }
  }
  if (saveEffPlot) {
    foodal->cd();
    weightHistogram->Draw("colz");
    foodal->SaveAs("plots/efficiency_bins.png");
    foodal->SetLogz(true);
    foodal->SaveAs("plots/efficiency_bins_log.png");
    foo->cd();
  }
 // Smooth
  Variable effSmoothing("effSmoothing", 0);
  SmoothHistogramPdf* ret = new SmoothHistogramPdf("efficiency", binEffData, effSmoothing);
  return ret;
}

GooPdf* makeBackgroundPdf () {
  vector<Observable> lvars;
  lvars.push_back(m12);
  lvars.push_back(m13);
  BinnedDataSet* binBkgData = new BinnedDataSet(lvars);
  createBackgroundHistogram();
  // Now testing your efficiency data by uniformly generating m12,m13 values
  TRandom3 donram(0);
  for (int i = 0; i < NevG; i++){
    do{
    m12.setValue(donram.Uniform(m12.getLowerLimit(), m12.getUpperLimit()));
    m13.setValue(donram.Uniform(m13.getLowerLimit(), m13.getUpperLimit()));
    }while(!cpuDalitz(m12.getValue(), m13.getValue(), MMass , D1Mass, D2Mass,D3Mass));
    //Weight will not be one if the physics boundary crosses the bin square.
    double weight = bkgHistogram->GetBinContent(bkgHistogram->FindBin(m12.getValue(), m13.getValue()));
    binBkgData->addWeightedEvent(weight);
    // Imposing the requirement on efficiency symmetry along m12=m13 line
      if (doEffSwap){
      double swapmass = m12.getValue(); m12.setValue(m13.getValue()); m13.setValue(swapmass);
      weight = bkgHistogram->GetBinContent(bkgHistogram->FindBin(m12.getValue(), m13.getValue()));
      binBkgData->addWeightedEvent(weight);
      }
  }
  if (saveBkgPlot) {
    foodal->cd();
    bkgHistogram->Draw("colz");
    foodal->SetLogz(false);
    foodal->SaveAs("plots/background_bins.png");
    foodal->SetLogz(true);
    foodal->SaveAs("plots/background_bins_log.png");
    foo->cd();
  }
  Variable* effSmoothing = new Variable("effSmoothing",0);
  SmoothHistogramPdf* ret = new SmoothHistogramPdf("efficiency", binBkgData, *effSmoothing);
  return ret;
}

vector<fptype> HH_bin_limits;
vector<Variable> pwa_coefs_amp;
vector<Variable> pwa_coefs_phs;

ResonancePdf* loadPWAResonance(const string fname = pwa_file, bool fixAmp = false,unsigned int cyc=PAIR_12){
  std::ifstream reader;
  reader.open(fname.c_str());
  assert(reader.good());
  HH_bin_limits.clear();
  pwa_coefs_amp.clear();
  pwa_coefs_phs.clear();
  double e1,e2,e3,e4;
  double emag,ephs;
  int i = 0;
  while (reader >> e1 >> e2 >> e3 >> e4) {
      HH_bin_limits.push_back(e1*e1);

      emag = sqrt(e2*e2+e3*e3);
      //emag = e2;
      ephs = TMath::ATan2(e3,e2);
      //ephs = e3;
      sprintf(strbuffer, "pwa_coef_%d_mag", i);
      Variable va(strbuffer, emag, .000001, 0, 10000);//0.9*emag, 1.1*emag);
      sprintf(strbuffer, "pwa_coef_%d_phase", i);
      Variable vp(strbuffer, ephs, .000001, -360, 360);//0.9*ephs, 1.1*ephs);

      pwa_coefs_amp.push_back(va);
      pwa_coefs_phs.push_back(vp);
      i++;
      cout << "s12 = " << e1*e1 << ", mag = " << emag << ", phs = " << (180/PI)*ephs << endl;
  }
  //const fptype scale = 1;
  Variable swave_amp_real("swave_amp_real", 3.0,   0.001, 0, 0);
  Variable swave_amp_imag("swave_amp_imag", 0.0,   0.001, 0, 0);
  swave_amp_real.setFixed(true);
  swave_amp_imag.setFixed(true);

  if (fixAmp) { swave_amp_real.setValue(1.); swave_amp_imag.setValue(0.); swave_amp_real.setFixed(true); swave_amp_imag.setFixed(true); }
  cout<<"Numbers loaded: "<<HH_bin_limits.size()<<" / "<<i<<endl;

  ResonancePdf* swave = new Resonances::Spline("swave", swave_amp_real,swave_amp_imag, HH_bin_limits, pwa_coefs_amp, pwa_coefs_phs,cyc);
  return swave;
}


DalitzPlotPdf* makeSignalPdf (GooPdf* eff,bool fixAmps) {
	DecayInfo3 dtop0pp;
	dtop0pp.motherMass  = MMass;
	dtop0pp.daug1Mass  = D1Mass;
	dtop0pp.daug2Mass  = D2Mass;
	dtop0pp.daug3Mass  = D3Mass;
	dtop0pp.meson_radius  = 1.5;


  /*  // Make a random number generater heres

		random_device rd;

		mt19937 mt(rd());

		normal_distribution<double> rand_gen(0.0,0.1);


	auto rhop  = new Resonances::RBW("rhop",
			Variable("rhop_amp_real", 1),
			Variable("rhop_amp_imag", 0),
			fixedRhoMass,
			fixedRhoWidth,
			1,
			PAIR_12);






    auto var_func = [&](std::string name, double start, double err) -> Variable {
       return fixAmps ?
              Variable(name, start) :
              Variable(name, start + rand_gen(mt), err, 0, 0);
    };*/



  //phi
  Variable phi_amp_real("phi_amp_real", 1);
  Variable phi_amp_imag("phi_amp_imag", 0);
  fixedPhiMass.setFixed(true);
  fixedPhiWidth.setFixed(true);

ResonancePdf* phi  = new Resonances::RBW("phi",phi_amp_real,phi_amp_imag,fixedPhiMass,fixedPhiWidth,1,PAIR_12,true);
ResonancePdf* phi13  = new Resonances::RBW("phi13",phi_amp_real,phi_amp_imag,fixedPhiMass,fixedPhiWidth,1,PAIR_13,true);


  // f0(980)
  Variable f0_amp_real("f0_amp_real",    12.341*cos(-62.852*(PI/180)),   0.0001, -100, 100);
  Variable f0_amp_imag("f0_amp_imag",    12.341*sin(-62.852*(PI/180)),   0.0001, -100, 100);
  Variable f0Mass("f0Mass", 0.965);
  Variable f0g1("f0g1", 0.165);
  Variable rg1og2("rg1og2", 4.21);//,1.0,5.0);

  ResonancePdf* f0  = new Resonances::FLATTE("f0",f0_amp_real,f0_amp_imag,f0Mass,f0g1,rg1og2,PAIR_12, true); //Required to be symmetric
	ResonancePdf* f013  = new Resonances::FLATTE("f013",f0_amp_real,f0_amp_imag,f0Mass,f0g1,rg1og2,PAIR_13, true); //Required to be symmetric

  // f0(X)

  Variable f0X_amp_real("f0X_amp_real",  11.918*cos(20.248*(PI/180)),   0.0001, -100, 100);
  Variable f0X_amp_imag("f0X_amp_imag",  11.918*sin(20.248*(PI/180)),   0.0001, -100, 100);
  Variable f0XMass("f0XMass",    1.41478);//,   0.00001,    1.00, 3.00);
  Variable f0XWidth("f0XWidth",  0.309491);//,   0.00001, 0.00005, 3.00);

  ResonancePdf* f0X  = new Resonances::RBW("f0X",f0X_amp_real,f0X_amp_imag,f0XMass,f0XWidth,(unsigned int)0,PAIR_12, true); //Required to be symmetric
	ResonancePdf* f0X13  = new Resonances::RBW("f0X13",f0X_amp_real,f0X_amp_imag,f0XMass,f0XWidth,(unsigned int)0,PAIR_13, true); //Required to be symmetric

  // NR
  Variable nonr_amp_real("nonr_amp_real", 0.0,   0.001, -100, +100);
  Variable nonr_amp_imag("nonr_amp_imag", 0.0,   0.001, -100, +100);
  ResonancePdf* nonr  = new Resonances::NonRes("nonr",nonr_amp_real,nonr_amp_imag);

  //bool fixAmps = false;
  ResonancePdf* swave = loadPWAResonance(pwa_file, fixAmps,PAIR_12);
  ResonancePdf* swave13 = loadPWAResonance(pwa_file, fixAmps,PAIR_13);

  dtop0pp.resonances.push_back(phi);
	dtop0pp.resonances.push_back(phi13);

	dtop0pp.resonances.push_back(swave);
	dtop0pp.resonances.push_back(swave13);

	//dtop0pp.resonances.push_back(f0X);
	//dtop0pp.resonances.push_back(f0X13);

	//dtop0pp.resonances.push_back(f0);
	//dtop0pp.resonances.push_back(f013);

	dtop0pp.resonances.push_back(nonr);




  if (!eff) {
    // By default create a constant efficiency.
    vector<Variable> offsets;
    vector<Observable> observables;
    vector<Variable> coefficients;

    observables.push_back(m12);
    observables.push_back(m13);
    offsets.push_back(constantZero);
    offsets.push_back(constantZero);
    coefficients.push_back(constantOne);
    eff = new PolynomialPdf("constantEff", observables, coefficients, offsets, 0);
  }
  comps.clear();

  return new DalitzPlotPdf("signalPDF", m12, m13, eventNumber, dtop0pp, eff);
}

void DalitzNorm(GooPdf* overallSignal,int N){


		random_device rd;
		mt19937 mt(rd());
		uniform_real_distribution<double> xyvalues(0.9,2.0);

		std::vector<Observable> vars;
		vars.push_back(m12);
		vars.push_back(m13);
		vars.push_back(eventNumber);

		std::vector<fptype> rpdfValuesvec;

    UnbinnedDataSet data(vars);
		eventNumber = 0;


		for(int i=0; i<N ; i++){

			m12 = xyvalues(mt);
			m13 = xyvalues(mt);



				if(cpuDalitz(m12.getValue(), m13.getValue())==1 ){
        	data.addEvent();

					eventNumber.setValue(eventNumber.getValue()+1);

				}else{

					data.addEvent();
					m12 =0;
					m13 =0;
					eventNumber.setValue(eventNumber.getValue()+1);
				}

		}



		overallSignal->setData(&data);
		signalDalitz->setDataSize(data.getNumEvents());
		std::vector<std::vector<double>> pdfValues = overallSignal->getCompProbsAtDataPoints();


		double buffer = 0;

		cout<< "Sample Size = " << pdfValues[0].size() << endl;

		 for(int k=0; k < pdfValues[0].size();k++){

			 buffer += pdfValues[0][k];

		}

		double  mean = buffer/N;
		double diff = 0;

		for(int l=0;l<pdfValues[0].size();l++){

			diff += (pdfValues[0][l] - mean)*(pdfValues[0][l] - mean);

		}

		double variance = diff/(N-1);
		double sigma = sqrt(variance);
		double RMS = sigma*V/sqrt(N);

		double integral = V*mean;

		std::cout << "Integral: "<< integral << "\t Error: " << RMS << "\n"; ;

}


void runIntegration(int N = 10000){

	  //TApplication* rootapp = new TApplication("rootapp",&argc,argv);

	signalDalitz = makeSignalPdf(0,false);

	std::vector<PdfBase*> comps;
	comps.clear();
	comps.push_back(signalDalitz);

	ProdPdf* overallSignal = new ProdPdf("overallSignal", comps);



	DalitzNorm(overallSignal,N);

	std::cout  << '\n';


}



void drawFitPlotsWithPulls(TH1* hd, TH1* ht, string plotdir){
	const char* hname = hd->GetName();
	char obsname[10];
	for (int i=0;;i++) {
		if (hname[i]=='_') obsname[i] = '\0';
		else obsname[i] = hname[i];
		if (obsname[i] == '\0') break;
	}
	ht->Scale(hd->Integral()/ht->Integral());
	foo->cd();
	foo->Clear();
	ht->Draw("l");
	hd->Draw("epsame");
	sprintf(strbuffer, "%s/%s_fit.png", plotdir.c_str(), obsname);
	foo->SaveAs(strbuffer);
	sprintf(strbuffer, "%s/%s_fit.pdf", plotdir.c_str(), obsname);
	foo->SaveAs(strbuffer);
	/*    sprintf(strbuffer, "%s/%s_fit_log.pdf", plotdir.c_str(), obsname);
		  foo->SaveAs(strbuffer);*/
}

void makeToyDalitzPdfPlots (GooPdf* overallSignal, string plotdir = "plots") {
	TH1F m12_dat_hist("m12_dat_hist", "", m12.getNumBins(), m12.getLowerLimit(), m12.getUpperLimit());
	m12_dat_hist.SetStats(false);
	m12_dat_hist.SetMarkerStyle(8);
	m12_dat_hist.SetMarkerSize(1);
	m12_dat_hist.GetXaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV]");
	sprintf(strbuffer, "Events / %.1f MeV", 1e3*m12_dat_hist.GetBinWidth(1));
	m12_dat_hist.GetYaxis()->SetTitle(strbuffer);
	TH1F m12_pdf_hist("m12_pdf_hist", "", m12.getNumBins(), m12.getLowerLimit(), m12.getUpperLimit());
	m12_pdf_hist.SetStats(false);
	m12_pdf_hist.SetLineColor(kBlue);
	m12_pdf_hist.SetLineWidth(3);
	TH1F m13_dat_hist("m13_dat_hist", "", m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());
	m13_dat_hist.SetStats(false);
	m13_dat_hist.SetMarkerStyle(8);
	m13_dat_hist.SetMarkerSize(1);
	m13_dat_hist.GetXaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV]");
	sprintf(strbuffer, "Events / %.1f MeV", 1e3*m13_dat_hist.GetBinWidth(1));
	m13_dat_hist.GetYaxis()->SetTitle(strbuffer);
	TH1F m13_pdf_hist("m13_pdf_hist", "", m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());
	m13_pdf_hist.SetStats(false);
	m13_pdf_hist.SetLineColor(kBlue);
	m13_pdf_hist.SetLineWidth(3);
	TH1F m23_dat_hist("m23_dat_hist", "", m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());
	m23_dat_hist.SetStats(false);
	m23_dat_hist.SetMarkerStyle(8);
	m23_dat_hist.SetMarkerSize(1.2);
	m23_dat_hist.GetXaxis()->SetTitle("m^{2}(#pi^{+} #pi^{-}) [GeV]");
	sprintf(strbuffer, "Events / %.1f MeV", 1e3*m13_dat_hist.GetBinWidth(1));
	m23_dat_hist.GetYaxis()->SetTitle(strbuffer);
	TH1F m23_pdf_hist("m23_pdf_hist", "", m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());
	m23_pdf_hist.SetStats(false);
	m23_pdf_hist.SetLineColor(kBlue);
	m23_pdf_hist.SetLineWidth(3);
	double totalPdf = 0;
	double totalDat = 0;
	TH2F dalitzpp0_dat_hist("dalitzpp0_dat_hist", "", m12.getNumBins(), m12.getLowerLimit(), m12.getUpperLimit(), m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());
	dalitzpp0_dat_hist.SetStats(false);
	dalitzpp0_dat_hist.GetXaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV]");
	dalitzpp0_dat_hist.GetYaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV^{2}]");
	TH2F dalitzpp0_pdf_hist("dalitzpp0_pdf_hist", "", m12.getNumBins(), m12.getLowerLimit(), m12.getUpperLimit(), m13.getNumBins(), m13.getLowerLimit(), m13.getUpperLimit());
	/*  dalitzpp0_pdf_hist.GetXaxis()->SetTitle("m^{2}(K^{-} #pi^{0}) [GeV^{2}]");
		dalitzpp0_pdf_hist.GetYaxis()->SetTitle("m^{2}(K^{-} #pi^{+}) [GeV^{2}]");*/
	dalitzpp0_pdf_hist.GetXaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV^{2}]");
	dalitzpp0_pdf_hist.GetYaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV^{2}]");
	dalitzpp0_pdf_hist.SetStats(false);
	std::vector<Observable> vars;
	vars.push_back(m12);
	vars.push_back(m13);
	vars.push_back(eventNumber);
	UnbinnedDataSet currData(vars);
	int evtCounter = 0;

	for (int i = 0; i < m12.getNumBins(); ++i) {
		m12.setValue(m12.getLowerLimit() + (m12.getUpperLimit() - m12.getLowerLimit())*(i + 0.5) / m12.getNumBins());
		for (int j = 0; j < m13.getNumBins(); ++j) {
			m13.setValue(m13.getLowerLimit() + (m13.getUpperLimit() - m13.getLowerLimit())*(j + 0.5) / m13.getNumBins());
			if (!cpuDalitz(m12.getValue(), m13.getValue(), MMass , D1Mass, D2Mass,D3Mass)) continue;
			eventNumber.setValue(evtCounter);
			evtCounter++;
			currData.addEvent();
		}
	}
	overallSignal->setData(&currData);
	signalDalitz->setDataSize(currData.getNumEvents());
	std::vector<std::vector<double> > pdfValues = overallSignal->getCompProbsAtDataPoints();
	for (unsigned int j = 0; j < pdfValues[0].size(); ++j) {
		double currm12 = currData.getValue(m12, j);
		double currm13 = currData.getValue(m13, j);

		dalitzpp0_pdf_hist.Fill(currm12, currm13, pdfValues[0][j]);
		m12_pdf_hist.Fill(currm12, pdfValues[0][j]);
		m13_pdf_hist.Fill(currm13, pdfValues[0][j]);
		m23_pdf_hist.Fill(cpuGetM23(currm12, currm13), pdfValues[0][j]);
		totalPdf     += pdfValues[0][j];
	}
	foodal->cd();
	foodal->SetLogz(false);
	dalitzpp0_pdf_hist.Draw("colz");
    std::string command = "mkdir -p " + plotdir;
    if (system(command.c_str()) != 0)
        throw GooFit::GeneralError("Making plot directory {} failed", plotdir);
	foodal->SaveAs((plotdir + "/dalitzpp0_pdf.png").c_str());
	/*  m12_pdf_hist.Draw("");
		foodal->SaveAs((plotdir + "/m12_pdf_hist.png").c_str());
		m13_pdf_hist.Draw("");
		foodal->SaveAs((plotdir + "/m13_pdf_hist.png").c_str());
		if (!data) return;*/
	for (unsigned int evt = 0; evt < data->getNumEvents(); ++evt) {
		double data_m12 = data->getValue(m12, evt);
		m12_dat_hist.Fill(data_m12);
		double data_m13 = data->getValue(m13, evt);
		m13_dat_hist.Fill(data_m13);
		dalitzpp0_dat_hist.Fill(data_m12, data_m13);
		m23_dat_hist.Fill(cpuGetM23(data_m12, data_m13));
		totalDat++;
	}
	dalitzpp0_dat_hist.Draw("colz");
	foodal->SaveAs((plotdir + "/dalitzpp0_dat.png").c_str());

	drawFitPlotsWithPulls(&m12_dat_hist, &m12_pdf_hist, plotdir);
	drawFitPlotsWithPulls(&m13_dat_hist, &m13_pdf_hist, plotdir);
	drawFitPlotsWithPulls(&m23_dat_hist, &m23_pdf_hist, plotdir);
}

void runToyFit (std::string toyFileName) {
	m12 = Observable("m12", 0.9, 2.0);
	m13 = Observable("m13", 0.9, 2.0);
	m12.setNumBins(nbins);
	m13.setNumBins(nbins);
	eventNumber = EventNumber("eventNumber", 0, INT_MAX);
	getToyData(toyFileName);


	signalDalitz = makeSignalPdf();
	comps.clear();
	comps.push_back(signalDalitz);
	ProdPdf* overallSignal = new ProdPdf("overallSignal", comps);
	overallSignal->setData(data);
	signalDalitz->setDataSize(data->getNumEvents());

	FitManager datapdf(overallSignal);

	for(int i=0;i<HH_bin_limits.size();i++){
      pwa_coefs_amp[i].setFixed(false);
      pwa_coefs_phs[i].setFixed(false);
      //pwa_coefs_amp[i]->error = pwa_coefs_phs[i]->error = 1.0;
  }

	gettimeofday(&startTime, NULL);
	startCPU = times(&startProc);
    datapdf.setVerbosity(verbosity);

		 // Maybe make optional? With a command line switch?
	datapdf.fit();
	stopCPU = times(&stopProc);
	gettimeofday(&stopTime, NULL);

	makeToyDalitzPdfPlots(overallSignal);
}

int main (int argc, char** argv) {

    GooFit::Application app{"D2K3_toy", argc, argv};
    app.add_option("-v,--verbose", verbosity, "Set the verbosity (to 0 for example", true);

    int fit_value;
    std::string name = "dalitz_mytoyMC_000.txt";

    auto fit = app.add_subcommand("fit");
    fit->add_option("-i,--int", fit_value, "A number to load");
    auto name_opt = fit->add_option("-n,--name,name", name, "The filename to load", true)
        ->excludes("--int");


		int N;
		auto run = app.add_subcommand("run");
		run->add_option("N",N, "")
		   ->required();


    int value;
    auto gen = app.add_subcommand("gen");
    gen->add_option("value", value, "The number to generate")
        ->required();

    app.require_subcommand(1);

    GOOFIT_PARSE(app);

    if(name_opt->count())
        name = fmt::format("dalitz_mytoyMC_{0:3}.txt", fit_value);

	gStyle->SetCanvasBorderMode(0);
	gStyle->SetCanvasColor(10);
	gStyle->SetFrameFillColor(10);
	gStyle->SetFrameBorderMode(0);
	gStyle->SetPadColor(0);
	gStyle->SetTitleColor(1);
	gStyle->SetStatColor(0);
	gStyle->SetFillColor(0);
	gStyle->SetFuncWidth(1);
	gStyle->SetLineWidth(1);
	gStyle->SetLineColor(1);
	gStyle->SetPalette(kViridis, 0);
	gStyle->SetNumberContours(512);
	gStyle->SetOptStat("RMe");
	foo = new TCanvas();
	foodal = new TCanvas();
	foodal->Size(10, 10);


    if(*fit)
	    runToyFit(name);
    if(*gen)
	    runToyGeneration(value);
		if(*run) {
				CLI::AutoTimer timer("Integration");
				runIntegration(N);
		}

	// Print total minimization time
	double myCPU = stopCPU - startCPU;
	double totalCPU = myCPU;

	timersub(&stopTime, &startTime, &totalTime);
	std::cout << "Wallclock time  : " << totalTime.tv_sec + totalTime.tv_usec/1000000.0 << " seconds." << std::endl;
	std::cout << "CPU time: " << (myCPU / CLOCKS_PER_SEC) << std::endl;
	std::cout << "Total CPU time: " << (totalCPU / CLOCKS_PER_SEC) << std::endl;
	myCPU = stopProc.tms_utime - startProc.tms_utime;
	std::cout << "Processor time: " << (myCPU / CLOCKS_PER_SEC) << std::endl;

	return 0;
}
